#include "hip/hip_runtime.h"
#include"gpuBruteForceSearch.cuh"



#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}




void bruteForceSearchAllAvailableFilesSeveralGpuLoadedEntrysAtATime(rangeSearchSettings& settings, ProteinDataHandler heldProteinSets, AtomToNumHashTable atomReferenceTable)
{

	int numberOfFilesInBatchPerRange[5];// = { 2, 2, 2, 2, 1 };
	loadMultiBatchRangeSizes("MultiBatchSizes.txt", settings);

	int requiredSizeForBatchesPerRange[5];
	for (int i = 0; i < 5; i++)
	{
		numberOfFilesInBatchPerRange[i] = settings.multiBatchRangeSizes[i];
		requiredSizeForBatchesPerRange[i] = numberOfFilesInBatchPerRange[i] * heldProteinSets.ProteinDataHolder[i].MaxEntrySize;
	}



	int largestRequiredBatchSizePos = largestSizeInArrayPos(requiredSizeForBatchesPerRange, 5);
	int largestCpuBatchSize = requiredSizeForBatchesPerRange[largestRequiredBatchSizePos];
	
	//all data has been preloaded into the host side ProteinDataHandler object. All that is needed is to initialise containers, moves sets of data to the gpu, process those sets of data and then return the results.
	std::cout << "PERFORMING TYPE 4 RANGE SEARCH: MULTI-BATCH GPU BRUTE FORCE" << std::endl;

	hipStream_t streams[2];
	hipStreamCreate(&streams[0]);
	hipStreamCreate(&streams[1]);

	//Initialise host and device data holders
	int SetBeingProcessed = 0;
	int SetBeingLoaded = 1;
	gpuBruteForceSingleEntryResources rangeSearchSlots[2];
	int IndividualEntryHolderSize = 16390 * 4;
	int BatchSetHolderSize =  largestCpuBatchSize; //The memory required to hold the largest batch of entriesrequired number of max size coordinate arrays + 6 more atoms
//	std::cout<<"Size of d_names in bytes: "<<BatchSetHolderSize* sizeof(short)<<std::endl;

	for (int i = 0; i < 2; i++)
	{
		rangeSearchSlots[i].h_resultsCount = (int*)malloc(1 * sizeof(int));
		rangeSearchSlots[i].h_resultsA = (int*)malloc(IndividualEntryHolderSize *10* sizeof(int));
		rangeSearchSlots[i].h_resultsB = (int*)malloc(IndividualEntryHolderSize *10* sizeof(int));
		rangeSearchSlots[i].h_aCount = (int*)malloc(1 * sizeof(int));
		rangeSearchSlots[i].h_bCount = (int*)malloc(1 * sizeof(int));
		rangeSearchSlots[i].h_elementAList = (int*)malloc(IndividualEntryHolderSize * sizeof(int)); //I used these for loading the element arrays back from the device to check what was in them, otherwise commented out.
		rangeSearchSlots[i].h_elementBList = (int*)malloc(IndividualEntryHolderSize * sizeof(int));
		rangeSearchSlots[i].threads = 512;
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_resultsCount, 1 * sizeof(int)));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_resultsA, IndividualEntryHolderSize *10* sizeof(int)));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_resultsB, IndividualEntryHolderSize *10* sizeof(int)));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_elementAList, IndividualEntryHolderSize * sizeof(int)));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_elementBList, IndividualEntryHolderSize * sizeof(int)));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_aCount, 1 * sizeof(int)));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_bCount, 1 * sizeof(int)));
		gpuErrchk(hipMemset(rangeSearchSlots[i].d_resultsCount, 0, 1 * sizeof(int)));
		gpuErrchk(hipMemset(rangeSearchSlots[i].d_resultsA, -1, IndividualEntryHolderSize *10* sizeof(int)));
		gpuErrchk(hipMemset(rangeSearchSlots[i].d_resultsB, -1, IndividualEntryHolderSize *10* sizeof(int)));
		gpuErrchk(hipMemset(rangeSearchSlots[i].d_aCount, 0, 1 * sizeof(int)));
		gpuErrchk(hipMemset(rangeSearchSlots[i].d_bCount, 0, 1 * sizeof(int)));

		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_names, BatchSetHolderSize* sizeof(short)*5 ));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_xCoords, BatchSetHolderSize* sizeof(int)*5));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_yCoords, BatchSetHolderSize* sizeof(int)*5));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_zCoords, BatchSetHolderSize* sizeof(int)*5));

	}


	int soughtAtomANumber = atomReferenceTable.retrieveHashValue(settings.AtomTypeOne);
	int soughtAtomBNumber = atomReferenceTable.retrieveHashValue(settings.AtomTypeTwo);

	int currentMaxEntrySize;
	int currentHeldEntries;


	for (int i = 0; i < 5; i++) //For each of the 5 range lengths of stored protein: Pin memory.
	{
		currentMaxEntrySize = heldProteinSets.ProteinDataHolder[i].MaxEntrySize;
		currentHeldEntries = heldProteinSets.ProteinDataHolder[i].heldEntries;

		if (currentHeldEntries > 0)
		{
			gpuErrchk(hipHostRegister(heldProteinSets.ProteinDataHolder[i].xCoordsSets, currentMaxEntrySize * currentHeldEntries * sizeof(int), 0));
			gpuErrchk(hipHostRegister(heldProteinSets.ProteinDataHolder[i].yCoordsSets, currentMaxEntrySize * currentHeldEntries * sizeof(int), 0));
			gpuErrchk(hipHostRegister(heldProteinSets.ProteinDataHolder[i].zCoordsSets, currentMaxEntrySize * currentHeldEntries * sizeof(int), 0));
			gpuErrchk(hipHostRegister(heldProteinSets.ProteinDataHolder[i].namesSets, currentMaxEntrySize * currentHeldEntries * sizeof(short), 0));
		}
	}

	for (int i = 0; i < 5; i++)//For each of the 5 range lengths of stored protein:
	{
//std::cout<<"Processing set: "<<i<<std::endl;
		//int temp;
		currentMaxEntrySize = heldProteinSets.ProteinDataHolder[i].MaxEntrySize;
		int TotalEntries = heldProteinSets.ProteinDataHolder[i].heldEntries;
		int NumberOfBatchesToProcess = TotalEntries / numberOfFilesInBatchPerRange[i]; //Needs to be rounded up - must check
		//if ((float(TotalEntries) / float(numberOfFilesInBatchPerRange[i])) > 0)
		//	NumberOfBatchesToProcess++;
		
		int sizeOfCurrentBatches;
		if (requiredSizeForBatchesPerRange[i]>=heldProteinSets.ProteinDataHolder[i].heldEntries)
			sizeOfCurrentBatches=requiredSizeForBatchesPerRange[i];
		else
			sizeOfCurrentBatches=heldProteinSets.ProteinDataHolder[i].heldEntries;
		
		clock_t n, m;

		if (currentMaxEntrySize < 513)
		{
			rangeSearchSlots[1].blocks = 1;
			rangeSearchSlots[0].blocks = 1;
		}
		else
		{
			rangeSearchSlots[1].blocks = (currentMaxEntrySize + rangeSearchSlots[1].threads - 1) / rangeSearchSlots[1].threads;
			rangeSearchSlots[0].blocks = (currentMaxEntrySize + rangeSearchSlots[1].threads - 1) / rangeSearchSlots[1].threads;
		}
		rangeSearchSlots[1].concurrentThreads = rangeSearchSlots[1].blocks*rangeSearchSlots[1].threads;
		rangeSearchSlots[0].concurrentThreads = rangeSearchSlots[0].blocks*rangeSearchSlots[0].threads;

		int outputType = settings.resultsPrintFormat;
		outputHandler filePrinter;
		std::string printType;
		if (outputType == 3) { printType = "_Summary"; }
		else if (outputType == 4) { printType = "_Detailed"; }

		if (outputType == 3 || outputType == 4)	{ filePrinter.initializeOutputfile("GpuBruteResults_Range_", currentMaxEntrySize, "_Files_", TotalEntries, printType); }

		n = clock();

		if (TotalEntries > 0)
		{
			std::cout << "Processing Range set: " << i << std::endl;
			std::cout << "Number of present entries is: " << heldProteinSets.ProteinDataHolder[i].heldEntries << std::endl;
			for (int currentBatch = 0; currentBatch < NumberOfBatchesToProcess + 1; currentBatch++)
			{
				//std::cout<<"processing batch: "<<currentBatch<<std::endl;
				if (currentBatch == 0)
				{
					
					//Load first set of details onto the gpu but do not process them -- needs work
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_resultsCount, 0, 1 * sizeof(int), streams[SetBeingLoaded]));
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_resultsA, -1, IndividualEntryHolderSize *10* sizeof(int), streams[SetBeingLoaded]));
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_resultsB, -1, IndividualEntryHolderSize *10* sizeof(int), streams[SetBeingLoaded]));
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_aCount, 0, 1 * sizeof(int), streams[SetBeingLoaded]));
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_bCount, 0, 1 * sizeof(int), streams[SetBeingLoaded]));
//std::cout<<std::endl<< "size of d_names"<<( BatchSetHolderSize* sizeof(int) / 2)<<std::endl<<"pos to copy from: "<< (sizeOfCurrentBatches * currentBatch) <<std::endl<<"size to copy: "<<(sizeOfCurrentBatches * sizeof(short))<<std::endl<<"First few name array elements:"<<std::endl<<heldProteinSets.ProteinDataHolder[i].namesSets[0]<<std::endl<<heldProteinSets.ProteinDataHolder[i].namesSets[1]<<std::endl<<heldProteinSets.ProteinDataHolder[i].namesSets[2]<<std::endl<<std::endl;
//for(int p=0;p<sizeOfCurrentBatches;p++)
//{
//std::cout<<heldProteinSets.ProteinDataHolder[i].namesSets[p]<<std::endl;
//}
//short temp[40000];
//for(int o=0;o<40000;i++)
//{
//temp[o]=o;
//}
					gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_names, /*temp*/heldProteinSets.ProteinDataHolder[i].namesSets + sizeOfCurrentBatches * currentBatch,sizeOfCurrentBatches * sizeof(short), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
				
///int test[500];
//for(int y=0;y<500;y++)
//{
//test[y]=y;
//}
//	gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_names,test,500*sizeof(int),hipMemcpyHostToDevice,streams[SetBeingLoaded]));
	gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_xCoords, heldProteinSets.ProteinDataHolder[i].xCoordsSets + sizeOfCurrentBatches * currentBatch, sizeOfCurrentBatches * sizeof(int), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
					gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_yCoords, heldProteinSets.ProteinDataHolder[i].yCoordsSets + sizeOfCurrentBatches * currentBatch, sizeOfCurrentBatches * sizeof(int), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
					gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_zCoords, heldProteinSets.ProteinDataHolder[i].zCoordsSets + sizeOfCurrentBatches * currentBatch, sizeOfCurrentBatches * sizeof(int), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
				}
				else if (currentBatch == NumberOfBatchesToProcess)
				{
					//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
					//process all proteins in loaded batch:

					for (int currentEntry = 0; currentEntry < numberOfFilesInBatchPerRange[i]; currentEntry++)
					{
						//current loaded protein
						rangeSearchSlots[SetBeingProcessed].h_aCount[0] = 0;
						rangeSearchSlots[SetBeingProcessed].h_bCount[0] = 0; //just for testing
						
						DeviceLoadedArrays_SingleProtein_LocateElements << <rangeSearchSlots[SetBeingProcessed].blocks, rangeSearchSlots[SetBeingProcessed].threads >> >(rangeSearchSlots[SetBeingProcessed].d_names, rangeSearchSlots[SetBeingProcessed].d_elementAList, rangeSearchSlots[SetBeingProcessed].d_elementBList, soughtAtomANumber, soughtAtomBNumber, rangeSearchSlots[SetBeingProcessed].d_aCount, rangeSearchSlots[SetBeingProcessed].d_bCount, currentMaxEntrySize, currentEntry, rangeSearchSlots[SetBeingProcessed].concurrentThreads, rangeSearchSlots[SetBeingProcessed].d_resultsA, rangeSearchSlots[SetBeingProcessed].d_resultsB, rangeSearchSlots[SetBeingProcessed].d_resultsCount, rangeSearchSlots[SetBeingProcessed].d_xCoords, rangeSearchSlots[SetBeingProcessed].d_yCoords, rangeSearchSlots[SetBeingProcessed].d_zCoords, settings.requiredProximity);
						DeviceLoadedArrays_SingleProtein_BruteForceSearch << <rangeSearchSlots[SetBeingProcessed].blocks, rangeSearchSlots[SetBeingProcessed].threads >> >(rangeSearchSlots[SetBeingProcessed].d_names, rangeSearchSlots[SetBeingProcessed].d_elementAList, rangeSearchSlots[SetBeingProcessed].d_elementBList, soughtAtomANumber, soughtAtomBNumber, rangeSearchSlots[SetBeingProcessed].d_aCount, rangeSearchSlots[SetBeingProcessed].d_bCount, currentMaxEntrySize, currentEntry, rangeSearchSlots[SetBeingProcessed].concurrentThreads, rangeSearchSlots[SetBeingProcessed].d_resultsA, rangeSearchSlots[SetBeingProcessed].d_resultsB, rangeSearchSlots[SetBeingProcessed].d_resultsCount, rangeSearchSlots[SetBeingProcessed].d_xCoords, rangeSearchSlots[SetBeingProcessed].d_yCoords, rangeSearchSlots[SetBeingProcessed].d_zCoords, settings.requiredProximity);

						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_elementAList, rangeSearchSlots[SetBeingProcessed].d_elementAList, currentMaxEntrySize * sizeof(int), hipMemcpyDeviceToHost));
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_elementBList, rangeSearchSlots[SetBeingProcessed].d_elementBList, currentMaxEntrySize * sizeof(int), hipMemcpyDeviceToHost));
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_aCount, rangeSearchSlots[SetBeingProcessed].d_aCount, 1 * sizeof(int), hipMemcpyDeviceToHost));
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_bCount, rangeSearchSlots[SetBeingProcessed].d_bCount, 1 * sizeof(int), hipMemcpyDeviceToHost));
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsCount, rangeSearchSlots[SetBeingProcessed].d_resultsCount, 1 * sizeof(int), hipMemcpyDeviceToHost));

						if (outputType == 1 || outputType == 2)
							std::cout << "Number of matches in file " << (currentEntry + (currentBatch-1)*numberOfFilesInBatchPerRange[i]) << " in set " << i << " is: " << rangeSearchSlots[SetBeingProcessed].h_resultsCount[0]/*h_aCount[0]*/ << std::endl;
						else if (outputType == 3 || outputType == 4)
							filePrinter.printLineToOutputFile("Number of matches in file ", (currentEntry + (currentBatch-1)*numberOfFilesInBatchPerRange[i]), " in set ", i, "  is: ", rangeSearchSlots[SetBeingProcessed].h_resultsCount[0]/*h_aCount[0]*/);

						//retrieve result arrays from device
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsCount, rangeSearchSlots[SetBeingProcessed].d_resultsCount, sizeof(int), hipMemcpyDeviceToHost));

						if (rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] < (currentMaxEntrySize * 9)) //If there are too many results, the program can fail. Not sure what the max limit on results (relative to reserved space) is though.
						{


							gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsA, rangeSearchSlots[SetBeingProcessed].d_resultsA, rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] * 1 * sizeof(int), hipMemcpyDeviceToHost));
							gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsB, rangeSearchSlots[SetBeingProcessed].d_resultsB, rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] * 1 * sizeof(int), hipMemcpyDeviceToHost));


							if (rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] > 0)
							{
								for (int j = 0; j < rangeSearchSlots[SetBeingProcessed].h_resultsCount[0]; j++)
								{
									if (outputType == 2)
									{
										std::cout << "AtomA: " << heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]] << "\t - Pos : " << rangeSearchSlots[SetBeingProcessed].h_resultsA[j] << "\t X: " << (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000) << "\t Y: " << (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000) << "\t Z: " << (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000) << std::endl;
										std::cout << "AtomB: " << heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]] << "\t - Pos : " << rangeSearchSlots[SetBeingProcessed].h_resultsB[j] << "\t X: " << (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000) << "\t Y: " << (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000) << "\t Z: " << (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000) << std::endl;
										std::cout << std::endl;
									}
									else if (outputType == 4)
									{
										filePrinter.printLineToOutputFile("Atom A: ", heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]], "\t - Pos : ", rangeSearchSlots[SetBeingProcessed].h_resultsA[j], "\t X: ", (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000), "\t Y: ", (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000), "\t Z: ", (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000));
										filePrinter.printLineToOutputFile("Atom B: ", heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]], "\t - Pos : ", rangeSearchSlots[SetBeingProcessed].h_resultsB[j], "\t X: ", (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000), "\t Y: ", (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000), "\t Z: ", (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000));
										filePrinter.printLineToOutputFile("");
									}

								}
							}

						}

						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingProcessed].d_resultsCount, 0, 1 * sizeof(int), streams[SetBeingProcessed]));
						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingProcessed].d_resultsA, -1, IndividualEntryHolderSize *10* sizeof(int), streams[SetBeingProcessed]));
						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingProcessed].d_resultsB, -1, IndividualEntryHolderSize *10* sizeof(int), streams[SetBeingProcessed]));
						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingProcessed].d_aCount, 0, 1 * sizeof(int), streams[SetBeingProcessed]));
						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingProcessed].d_bCount, 0, 1 * sizeof(int), streams[SetBeingProcessed]));

					}

					//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


				}
				else
				{


					//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
					//Load next batch of entries onto the gpu.
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_resultsCount, 0, 1 * sizeof(int), streams[SetBeingLoaded]));
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_resultsA, -1, IndividualEntryHolderSize *10* sizeof(int), streams[SetBeingLoaded]));
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_resultsB, -1, IndividualEntryHolderSize*10* sizeof(int), streams[SetBeingLoaded]));
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_aCount, 0, 1 * sizeof(int), streams[SetBeingLoaded]));
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_bCount, 0, 1 * sizeof(int), streams[SetBeingLoaded]));
					gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_names, heldProteinSets.ProteinDataHolder[i].namesSets + sizeOfCurrentBatches * currentBatch, sizeOfCurrentBatches * sizeof(short), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
					gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_xCoords, heldProteinSets.ProteinDataHolder[i].xCoordsSets + sizeOfCurrentBatches * currentBatch, sizeOfCurrentBatches * sizeof(int), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
					gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_yCoords, heldProteinSets.ProteinDataHolder[i].yCoordsSets + sizeOfCurrentBatches * currentBatch, sizeOfCurrentBatches * sizeof(int), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
					gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_zCoords, heldProteinSets.ProteinDataHolder[i].zCoordsSets + sizeOfCurrentBatches * currentBatch, sizeOfCurrentBatches * sizeof(int), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
					//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////



					//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
					//process all proteins in loaded batch:

					for (int currentEntry = 0; currentEntry < numberOfFilesInBatchPerRange[i]; currentEntry++)
					{
						//current loaded protein
						rangeSearchSlots[SetBeingProcessed].h_aCount[0] = 0;
						rangeSearchSlots[SetBeingProcessed].h_bCount[0] = 0; //just for testing


						DeviceLoadedArrays_SingleProtein_LocateElements << <rangeSearchSlots[SetBeingProcessed].blocks, rangeSearchSlots[SetBeingProcessed].threads >> >(rangeSearchSlots[SetBeingProcessed].d_names, rangeSearchSlots[SetBeingProcessed].d_elementAList, rangeSearchSlots[SetBeingProcessed].d_elementBList, soughtAtomANumber, soughtAtomBNumber, rangeSearchSlots[SetBeingProcessed].d_aCount, rangeSearchSlots[SetBeingProcessed].d_bCount, currentMaxEntrySize, currentEntry, rangeSearchSlots[SetBeingProcessed].concurrentThreads, rangeSearchSlots[SetBeingProcessed].d_resultsA, rangeSearchSlots[SetBeingProcessed].d_resultsB, rangeSearchSlots[SetBeingProcessed].d_resultsCount, rangeSearchSlots[SetBeingProcessed].d_xCoords, rangeSearchSlots[SetBeingProcessed].d_yCoords, rangeSearchSlots[SetBeingProcessed].d_zCoords, settings.requiredProximity);
						DeviceLoadedArrays_SingleProtein_BruteForceSearch << <rangeSearchSlots[SetBeingProcessed].blocks, rangeSearchSlots[SetBeingProcessed].threads >> >(rangeSearchSlots[SetBeingProcessed].d_names, rangeSearchSlots[SetBeingProcessed].d_elementAList, rangeSearchSlots[SetBeingProcessed].d_elementBList, soughtAtomANumber, soughtAtomBNumber, rangeSearchSlots[SetBeingProcessed].d_aCount, rangeSearchSlots[SetBeingProcessed].d_bCount, currentMaxEntrySize, currentEntry, rangeSearchSlots[SetBeingProcessed].concurrentThreads, rangeSearchSlots[SetBeingProcessed].d_resultsA, rangeSearchSlots[SetBeingProcessed].d_resultsB, rangeSearchSlots[SetBeingProcessed].d_resultsCount, rangeSearchSlots[SetBeingProcessed].d_xCoords, rangeSearchSlots[SetBeingProcessed].d_yCoords, rangeSearchSlots[SetBeingProcessed].d_zCoords, settings.requiredProximity);

						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_elementAList, rangeSearchSlots[SetBeingProcessed].d_elementAList, currentMaxEntrySize * sizeof(int), hipMemcpyDeviceToHost));
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_elementBList, rangeSearchSlots[SetBeingProcessed].d_elementBList, currentMaxEntrySize * sizeof(int), hipMemcpyDeviceToHost));
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_aCount, rangeSearchSlots[SetBeingProcessed].d_aCount, 1 * sizeof(int), hipMemcpyDeviceToHost));
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_bCount, rangeSearchSlots[SetBeingProcessed].d_bCount, 1 * sizeof(int), hipMemcpyDeviceToHost));
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsCount, rangeSearchSlots[SetBeingProcessed].d_resultsCount, 1 * sizeof(int), hipMemcpyDeviceToHost));

						if (outputType == 1 || outputType == 2)
							std::cout << "Number of matches in file " << (currentEntry + (currentBatch-1)*numberOfFilesInBatchPerRange[i]) << " in set " << i << " is: " << rangeSearchSlots[SetBeingProcessed].h_resultsCount[0]/*h_aCount[0]*/ << std::endl;
						else if (outputType == 3 || outputType == 4)
							filePrinter.printLineToOutputFile("Number of matches in file ", (currentEntry + (currentBatch-1)*numberOfFilesInBatchPerRange[i]), " in set ", i, "  is: ", rangeSearchSlots[SetBeingProcessed].h_resultsCount[0]/*h_aCount[0]*/);

						//retrieve result arrays from device
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsCount, rangeSearchSlots[SetBeingProcessed].d_resultsCount, sizeof(int), hipMemcpyDeviceToHost));


						if (rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] < (currentMaxEntrySize * 9)) //If there are too many results, the program can fail. Not sure what the max limit on results (relative to reserved space) is though.
						{


							gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsA, rangeSearchSlots[SetBeingProcessed].d_resultsA, rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] * 1 * sizeof(int), hipMemcpyDeviceToHost));
							gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsB, rangeSearchSlots[SetBeingProcessed].d_resultsB, rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] * 1 * sizeof(int), hipMemcpyDeviceToHost));


							if (rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] > 0)
							{
								for (int j = 0; j < rangeSearchSlots[SetBeingProcessed].h_resultsCount[0]; j++)
								{
									if (outputType == 2)
									{
										std::cout << "AtomA: " << heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]] << "\t - Pos : " << rangeSearchSlots[SetBeingProcessed].h_resultsA[j] << "\t X: " << (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000) << "\t Y: " << (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000) << "\t Z: " << (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000) << std::endl;
										std::cout << "AtomB: " << heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]] << "\t - Pos : " << rangeSearchSlots[SetBeingProcessed].h_resultsB[j] << "\t X: " << (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000) << "\t Y: " << (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000) << "\t Z: " << (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000) << std::endl;
										std::cout << std::endl;
									}
									else if (outputType == 4)
									{
										filePrinter.printLineToOutputFile("Atom A: ", heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]], "\t - Pos : ", rangeSearchSlots[SetBeingProcessed].h_resultsA[j], "\t X: ", (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000), "\t Y: ", (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000), "\t Z: ", (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000));
										filePrinter.printLineToOutputFile("Atom B: ", heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]], "\t - Pos : ", rangeSearchSlots[SetBeingProcessed].h_resultsB[j], "\t X: ", (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000), "\t Y: ", (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000), "\t Z: ", (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000));
										filePrinter.printLineToOutputFile("");
									}

								}
							}

						}

						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingProcessed].d_resultsCount, 0, 1 * sizeof(int), streams[SetBeingProcessed]));
						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingProcessed].d_resultsA, -1, IndividualEntryHolderSize *10* sizeof(int), streams[SetBeingProcessed]));
						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingProcessed].d_resultsB, -1, IndividualEntryHolderSize *10* sizeof(int), streams[SetBeingProcessed]));
						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingProcessed].d_aCount, 0, 1 * sizeof(int), streams[SetBeingProcessed]));
						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingProcessed].d_bCount, 0, 1 * sizeof(int), streams[SetBeingProcessed]));
					}
					//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


				}

				//swap the pointer to which set needs to be processed and loaded next - so that the previously loaded set will be processed and a new set will be loaded into the spot of the already loaded set.
				switchLoadingAndProcessingSets(SetBeingProcessed, SetBeingLoaded);


				//run bruteForce set runner



			}
			m = clock();
			print_elapsed(n, m, "run time for bruteset mini: ");
			std::cout << std::endl;
		}

		filePrinter.closeOpenFile();
		
		
	}




	for (int i = 0; i < 5; i++) //For each of the 5 range lengths of stored protein: Unpin memory.
	{
		currentHeldEntries = heldProteinSets.ProteinDataHolder[i].heldEntries;

		if (currentHeldEntries > 0)
		{
			gpuErrchk(hipHostUnregister(heldProteinSets.ProteinDataHolder[i].xCoordsSets));
			gpuErrchk(hipHostUnregister(heldProteinSets.ProteinDataHolder[i].yCoordsSets));
			gpuErrchk(hipHostUnregister(heldProteinSets.ProteinDataHolder[i].zCoordsSets));
			gpuErrchk(hipHostUnregister(heldProteinSets.ProteinDataHolder[i].namesSets));
		}
	}

	for (int i = 0; i < 2; i++)
	{
		gpuErrchk(hipFree(rangeSearchSlots[i].d_xCoords));
		gpuErrchk(hipFree(rangeSearchSlots[i].d_yCoords));
		gpuErrchk(hipFree(rangeSearchSlots[i].d_zCoords));
		gpuErrchk(hipFree(rangeSearchSlots[i].d_names));

		gpuErrchk(hipFree(rangeSearchSlots[i].d_resultsCount));
		gpuErrchk(hipFree(rangeSearchSlots[i].d_resultsA));
		gpuErrchk(hipFree(rangeSearchSlots[i].d_resultsB));
		gpuErrchk(hipFree(rangeSearchSlots[i].d_elementAList));
		gpuErrchk(hipFree(rangeSearchSlots[i].d_elementBList));
		gpuErrchk(hipFree(rangeSearchSlots[i].d_aCount));
		gpuErrchk(hipFree(rangeSearchSlots[i].d_bCount));
	}
	/*gpuErrchk(hipFree(rangeSearchSlots[1].d_xCoords));
	gpuErrchk(hipFree(rangeSearchSlots[1].d_yCoords));
	gpuErrchk(hipFree(rangeSearchSlots[1].d_zCoords));
	gpuErrchk(hipFree(rangeSearchSlots[1].d_names));*/

	hipStreamDestroy(streams[0]);
	hipStreamDestroy(streams[1]);

hipDeviceReset(); 
}

void bruteForceSearchAllAvailableFilesSingleGpuLoadedEntryAtATime(rangeSearchSettings& settings, ProteinDataHandler heldProteinSets, AtomToNumHashTable atomReferenceTable)
{
	//all data has been preloaded into the host side ProteinDataHandler object. All that is needed is to initialise containers, moves sets of data to the gpu, process those sets of data and then return the results.
	std::cout << "PERFORMING TYPE 3 RANGE SEARCH: INDIVIDUAL LOAD GPU BRUTE FORCE RANGE SEARCH" << std::endl;

	hipStream_t streams[2];
	hipStreamCreate(&streams[0]);
	hipStreamCreate(&streams[1]);

	//Initialise host and device data holders
	int SetBeingProcessed = 0;
	int SetBeingLoaded = 1;
	gpuBruteForceSingleEntryResources rangeSearchSlots[2];
	//gpuBruteForceSingleEntryResources rangeSearchSlotB;






	int IndividualEntryHolderSize = 16390 * 5; //The memory required to hold a max size coordinate array + 6 more atoms

	for (int i = 0; i < 2; i++)
	{
		rangeSearchSlots[i].h_resultsCount = (int*)malloc(1 * sizeof(int));
		rangeSearchSlots[i].h_resultsA = (int*)malloc(IndividualEntryHolderSize * sizeof(int));
		rangeSearchSlots[i].h_resultsB = (int*)malloc(IndividualEntryHolderSize * sizeof(int));
		rangeSearchSlots[i].h_aCount = (int*)malloc(1 * sizeof(int));
		rangeSearchSlots[i].h_bCount = (int*)malloc(1 * sizeof(int));
		rangeSearchSlots[i].h_elementAList = (int*)malloc(IndividualEntryHolderSize * sizeof(int)); //I used these for loading the element arrays back from the device to check what was in them, otherwise commented out.
		rangeSearchSlots[i].h_elementBList = (int*)malloc(IndividualEntryHolderSize * sizeof(int));
		rangeSearchSlots[i].threads = 512;
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_resultsCount, 1 * sizeof(int)));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_resultsA, IndividualEntryHolderSize * sizeof(int)));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_resultsB, IndividualEntryHolderSize * sizeof(int)));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_elementAList, IndividualEntryHolderSize  * sizeof(int)));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_elementBList, IndividualEntryHolderSize  * sizeof(int)));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_aCount, 1 * sizeof(int)));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_bCount, 1 * sizeof(int)));
		gpuErrchk(hipMemset(rangeSearchSlots[i].d_resultsCount, 0, 1 * sizeof(int)));
		gpuErrchk(hipMemset(rangeSearchSlots[i].d_resultsA, -1, IndividualEntryHolderSize  * sizeof(int)));
		gpuErrchk(hipMemset(rangeSearchSlots[i].d_resultsB, -1, IndividualEntryHolderSize  * sizeof(int)));
		gpuErrchk(hipMemset(rangeSearchSlots[i].d_aCount, 0, 1 * sizeof(int)));
		gpuErrchk(hipMemset(rangeSearchSlots[i].d_bCount, 0, 1 * sizeof(int)));

		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_names, IndividualEntryHolderSize / 2));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_xCoords, IndividualEntryHolderSize));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_yCoords, IndividualEntryHolderSize));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_zCoords, IndividualEntryHolderSize));

	}

	int soughtAtomANumber = atomReferenceTable.retrieveHashValue(settings.AtomTypeOne);
	int soughtAtomBNumber = atomReferenceTable.retrieveHashValue(settings.AtomTypeTwo);

	int currentMaxEntrySize;
	int currentHeldEntries;


	for (int i = 0; i < 5; i++) //For each of the 5 range lengths of stored protein: Pin memory.
	{
		currentMaxEntrySize = heldProteinSets.ProteinDataHolder[i].MaxEntrySize;
		currentHeldEntries = heldProteinSets.ProteinDataHolder[i].heldEntries;
		if (currentHeldEntries > 0)
		{
			gpuErrchk(hipHostRegister(heldProteinSets.ProteinDataHolder[i].xCoordsSets, currentMaxEntrySize * currentHeldEntries * sizeof(int), 0));
			gpuErrchk(hipHostRegister(heldProteinSets.ProteinDataHolder[i].yCoordsSets, currentMaxEntrySize * currentHeldEntries * sizeof(int), 0));
			gpuErrchk(hipHostRegister(heldProteinSets.ProteinDataHolder[i].zCoordsSets, currentMaxEntrySize * currentHeldEntries * sizeof(int), 0));
			gpuErrchk(hipHostRegister(heldProteinSets.ProteinDataHolder[i].namesSets, currentMaxEntrySize * currentHeldEntries * sizeof(short), 0));
		}
	}

	for (int i = 0; i < 5; i++)//For each of the 5 range lengths of stored protein:
	{
	//	std::cout<<"Processing set: "<<i<<std::endl;
		currentMaxEntrySize = heldProteinSets.ProteinDataHolder[i].MaxEntrySize;
		int TotalEntries = heldProteinSets.ProteinDataHolder[i].heldEntries;
		
		clock_t n, m;

		if (currentMaxEntrySize < 513)
		{
			rangeSearchSlots[1].blocks = 1;
			rangeSearchSlots[0].blocks = 1;
		}
		else
		{
			rangeSearchSlots[1].blocks = (currentMaxEntrySize + rangeSearchSlots[1].threads - 1) / rangeSearchSlots[1].threads;
			rangeSearchSlots[0].blocks = (currentMaxEntrySize + rangeSearchSlots[1].threads - 1) / rangeSearchSlots[1].threads;
		}
		rangeSearchSlots[1].concurrentThreads = rangeSearchSlots[1].blocks*rangeSearchSlots[1].threads;
		rangeSearchSlots[0].concurrentThreads = rangeSearchSlots[0].blocks*rangeSearchSlots[0].threads;

		int outputType = settings.resultsPrintFormat;
		outputHandler filePrinter;
		std::string printType;
		if (outputType == 3)
			printType = "_Summary"; 
		else if (outputType == 4) 
			printType = "_Detailed"; 

		if (outputType == 3 || outputType == 4)	{ filePrinter.initializeOutputfile("GpuBruteResults_Range_", currentMaxEntrySize, "_Files_", TotalEntries, printType); }

		n = clock();

		if (TotalEntries > 0)
		{
			std::cout << "Processing Range set: " << i << std::endl;
			std::cout << "Number of present entries is: " << heldProteinSets.ProteinDataHolder[i].heldEntries << std::endl;
			for (int currentEntry = 0; currentEntry < TotalEntries + 1; currentEntry++)
			{
				//std::cout<<"Processing entry: "<<currentEntry<<"entrySize: "<<heldProteinSets.ProteinDataHolder[i].proteinLengthCounts[currentEntry]<<std::endl;
				if (currentEntry == 0)
				{

					//Load first set of details onto the gpu but do not process them -- needs work
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_resultsCount, 0, 1 * sizeof(int), streams[SetBeingLoaded]));
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_resultsA, -1, IndividualEntryHolderSize * sizeof(int), streams[SetBeingLoaded]));
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_resultsB, -1, IndividualEntryHolderSize * sizeof(int), streams[SetBeingLoaded]));
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_aCount, 0, 1 * sizeof(int), streams[SetBeingLoaded]));
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_bCount, 0, 1 * sizeof(int), streams[SetBeingLoaded]));
					gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_names, heldProteinSets.ProteinDataHolder[i].namesSets + currentMaxEntrySize*currentEntry, currentMaxEntrySize * sizeof(short), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
					gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_xCoords, heldProteinSets.ProteinDataHolder[i].xCoordsSets + currentMaxEntrySize*currentEntry, currentMaxEntrySize *  sizeof(int), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
					gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_yCoords, heldProteinSets.ProteinDataHolder[i].yCoordsSets + currentMaxEntrySize*currentEntry, currentMaxEntrySize * sizeof(int), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
					gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_zCoords, heldProteinSets.ProteinDataHolder[i].zCoordsSets + currentMaxEntrySize*currentEntry, currentMaxEntrySize * sizeof(int), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
				}
				else if (currentEntry == TotalEntries)
				{
					//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
					//process current loaded set
					rangeSearchSlots[SetBeingProcessed].h_aCount[0] = 0;
					rangeSearchSlots[SetBeingProcessed].h_bCount[0] = 0; //just for testing




					DeviceLoadedArrays_SingleProtein_LocateElements << <rangeSearchSlots[SetBeingProcessed].blocks, rangeSearchSlots[SetBeingProcessed].threads >> >(rangeSearchSlots[SetBeingProcessed].d_names, rangeSearchSlots[SetBeingProcessed].d_elementAList, rangeSearchSlots[SetBeingProcessed].d_elementBList, soughtAtomANumber, soughtAtomBNumber, rangeSearchSlots[SetBeingProcessed].d_aCount, rangeSearchSlots[SetBeingProcessed].d_bCount, currentMaxEntrySize, 0, rangeSearchSlots[SetBeingProcessed].concurrentThreads, rangeSearchSlots[SetBeingProcessed].d_resultsA, rangeSearchSlots[SetBeingProcessed].d_resultsB, rangeSearchSlots[SetBeingProcessed].d_resultsCount, rangeSearchSlots[SetBeingProcessed].d_xCoords, rangeSearchSlots[SetBeingProcessed].d_yCoords, rangeSearchSlots[SetBeingProcessed].d_zCoords, settings.requiredProximity);
					DeviceLoadedArrays_SingleProtein_BruteForceSearch << <rangeSearchSlots[SetBeingProcessed].blocks, rangeSearchSlots[SetBeingProcessed].threads >> >(rangeSearchSlots[SetBeingProcessed].d_names, rangeSearchSlots[SetBeingProcessed].d_elementAList, rangeSearchSlots[SetBeingProcessed].d_elementBList, soughtAtomANumber, soughtAtomBNumber, rangeSearchSlots[SetBeingProcessed].d_aCount, rangeSearchSlots[SetBeingProcessed].d_bCount, currentMaxEntrySize, 0, rangeSearchSlots[SetBeingProcessed].concurrentThreads, rangeSearchSlots[SetBeingProcessed].d_resultsA, rangeSearchSlots[SetBeingProcessed].d_resultsB, rangeSearchSlots[SetBeingProcessed].d_resultsCount, rangeSearchSlots[SetBeingProcessed].d_xCoords, rangeSearchSlots[SetBeingProcessed].d_yCoords, rangeSearchSlots[SetBeingProcessed].d_zCoords, settings.requiredProximity);

					gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_elementAList, rangeSearchSlots[SetBeingProcessed].d_elementAList, currentMaxEntrySize * sizeof(int), hipMemcpyDeviceToHost));
					gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_elementBList, rangeSearchSlots[SetBeingProcessed].d_elementBList, currentMaxEntrySize * sizeof(int), hipMemcpyDeviceToHost));
					gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_aCount, rangeSearchSlots[SetBeingProcessed].d_aCount, 1 * sizeof(int), hipMemcpyDeviceToHost));
					gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_bCount, rangeSearchSlots[SetBeingProcessed].d_bCount, 1 * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsCount, rangeSearchSlots[SetBeingProcessed].d_resultsCount, 1 * sizeof(int), hipMemcpyDeviceToHost));


					if (outputType == 1 || outputType == 2)
						std::cout << "Number of matches in file " << currentEntry << " in set " << i << " is: " << rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] << std::endl;
					else if (outputType == 3 || outputType == 4)
						filePrinter.printLineToOutputFile("Number of matches in file ", currentEntry, " in set ", i, "  is: ", rangeSearchSlots[SetBeingProcessed].h_resultsCount[0]);

					//retrieve result arrays from device
					gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsCount, rangeSearchSlots[SetBeingProcessed].d_resultsCount, sizeof(int), hipMemcpyDeviceToHost));


					if (rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] < (currentMaxEntrySize * 9)) //If there are too many results, the program can fail. Not sure what the max limit on results (relative to reserved space) is though.
					{


						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsA, rangeSearchSlots[SetBeingProcessed].d_resultsA, rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] * 1 * sizeof(int), hipMemcpyDeviceToHost));
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsB, rangeSearchSlots[SetBeingProcessed].d_resultsB, rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] * 1 * sizeof(int), hipMemcpyDeviceToHost));


						if (rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] > 0)
						{
							for (int j = 0; j < rangeSearchSlots[SetBeingProcessed].h_resultsCount[0]; j++)
							{
								if (outputType == 2)
								{
									std::cout << "AtomA: " << heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]] << "\t - Pos : " << rangeSearchSlots[SetBeingProcessed].h_resultsA[j] << "\t X: " << (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000) << "\t Y: " << (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000) << "\t Z: " << (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000) << std::endl;
									std::cout << "AtomB: " << heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]] << "\t - Pos : " << rangeSearchSlots[SetBeingProcessed].h_resultsB[j] << "\t X: " << (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000) << "\t Y: " << (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000) << "\t Z: " << (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000) << std::endl;
									std::cout << std::endl;
								}
								else if (outputType == 4)
								{
									filePrinter.printLineToOutputFile("Atom A: ", heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]], "\t - Pos : ", rangeSearchSlots[SetBeingProcessed].h_resultsA[j], "\t X: ", (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000), "\t Y: ", (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000), "\t Z: ", (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000));
									filePrinter.printLineToOutputFile("Atom B: ", heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]], "\t - Pos : ", rangeSearchSlots[SetBeingProcessed].h_resultsB[j], "\t X: ", (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000), "\t Y: ", (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000), "\t Z: ", (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000));
									filePrinter.printLineToOutputFile("");
								}

							}
						}

					}
					//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


				}
				else
				{


					//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
					//Load next set
					//Load next set of details onto the gpu 
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_resultsCount, 0, 1 * sizeof(int), streams[SetBeingLoaded]));
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_resultsA, -1, IndividualEntryHolderSize * sizeof(int), streams[SetBeingLoaded]));
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_resultsB, -1, IndividualEntryHolderSize * sizeof(int), streams[SetBeingLoaded]));
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_aCount, 0, 1 * sizeof(int), streams[SetBeingLoaded]));
					gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_bCount, 0, 1 * sizeof(int), streams[SetBeingLoaded]));
					gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_names, heldProteinSets.ProteinDataHolder[i].namesSets + currentMaxEntrySize*currentEntry, currentMaxEntrySize * sizeof(short), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
					gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_xCoords, heldProteinSets.ProteinDataHolder[i].xCoordsSets + currentMaxEntrySize*currentEntry, currentMaxEntrySize *  sizeof(int), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
					gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_yCoords, heldProteinSets.ProteinDataHolder[i].yCoordsSets + currentMaxEntrySize*currentEntry, currentMaxEntrySize * sizeof(int), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
					gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_zCoords, heldProteinSets.ProteinDataHolder[i].zCoordsSets + currentMaxEntrySize*currentEntry, currentMaxEntrySize * sizeof(int), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
					//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////



					//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
					//process current loaded set
					rangeSearchSlots[SetBeingProcessed].h_aCount[0] = 0;
					rangeSearchSlots[SetBeingProcessed].h_bCount[0] = 0; //just for testing


					DeviceLoadedArrays_SingleProtein_LocateElements << <rangeSearchSlots[SetBeingProcessed].blocks, rangeSearchSlots[SetBeingProcessed].threads >> >(rangeSearchSlots[SetBeingProcessed].d_names, rangeSearchSlots[SetBeingProcessed].d_elementAList, rangeSearchSlots[SetBeingProcessed].d_elementBList, soughtAtomANumber, soughtAtomBNumber, rangeSearchSlots[SetBeingProcessed].d_aCount, rangeSearchSlots[SetBeingProcessed].d_bCount, currentMaxEntrySize, 0, rangeSearchSlots[SetBeingProcessed].concurrentThreads, rangeSearchSlots[SetBeingProcessed].d_resultsA, rangeSearchSlots[SetBeingProcessed].d_resultsB, rangeSearchSlots[SetBeingProcessed].d_resultsCount, rangeSearchSlots[SetBeingProcessed].d_xCoords, rangeSearchSlots[SetBeingProcessed].d_yCoords, rangeSearchSlots[SetBeingProcessed].d_zCoords, settings.requiredProximity);
					DeviceLoadedArrays_SingleProtein_BruteForceSearch << <rangeSearchSlots[SetBeingProcessed].blocks, rangeSearchSlots[SetBeingProcessed].threads >> >(rangeSearchSlots[SetBeingProcessed].d_names, rangeSearchSlots[SetBeingProcessed].d_elementAList, rangeSearchSlots[SetBeingProcessed].d_elementBList, soughtAtomANumber, soughtAtomBNumber, rangeSearchSlots[SetBeingProcessed].d_aCount, rangeSearchSlots[SetBeingProcessed].d_bCount, currentMaxEntrySize, 0, rangeSearchSlots[SetBeingProcessed].concurrentThreads, rangeSearchSlots[SetBeingProcessed].d_resultsA, rangeSearchSlots[SetBeingProcessed].d_resultsB, rangeSearchSlots[SetBeingProcessed].d_resultsCount, rangeSearchSlots[SetBeingProcessed].d_xCoords, rangeSearchSlots[SetBeingProcessed].d_yCoords, rangeSearchSlots[SetBeingProcessed].d_zCoords, settings.requiredProximity);

					gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_elementAList, rangeSearchSlots[SetBeingProcessed].d_elementAList, currentMaxEntrySize * sizeof(int), hipMemcpyDeviceToHost));
					gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_elementBList, rangeSearchSlots[SetBeingProcessed].d_elementBList, currentMaxEntrySize * sizeof(int), hipMemcpyDeviceToHost));
					gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_aCount, rangeSearchSlots[SetBeingProcessed].d_aCount, 1 * sizeof(int), hipMemcpyDeviceToHost));
					gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_bCount, rangeSearchSlots[SetBeingProcessed].d_bCount, 1 * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsCount, rangeSearchSlots[SetBeingProcessed].d_resultsCount, 1 * sizeof(int), hipMemcpyDeviceToHost));


					if (outputType == 1 || outputType == 2)
						std::cout << "Number of matches in file " << currentEntry << " in set " << i << " is: " << rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] << std::endl;
					else if (outputType == 3 || outputType == 4)
						filePrinter.printLineToOutputFile("Number of matches in file ", currentEntry, " in set ", i, "  is: ", rangeSearchSlots[SetBeingProcessed].h_resultsCount[0]);

					//retrieve result arrays from device
					gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsCount, rangeSearchSlots[SetBeingProcessed].d_resultsCount, sizeof(int), hipMemcpyDeviceToHost));


					if (rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] < (currentMaxEntrySize * 9)) //If there are too many results, the program can fail. Not sure what the max limit on results (relative to reserved space) is though.
					{


						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsA, rangeSearchSlots[SetBeingProcessed].d_resultsA, rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] * 1 * sizeof(int), hipMemcpyDeviceToHost));
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsB, rangeSearchSlots[SetBeingProcessed].d_resultsB, rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] * 1 * sizeof(int), hipMemcpyDeviceToHost));


						if (rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] > 0)
						{
							for (int j = 0; j < rangeSearchSlots[SetBeingProcessed].h_resultsCount[0]; j++)
							{
								if (outputType == 2)
								{
									std::cout << "AtomA: " << heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]] << "\t - Pos : " << rangeSearchSlots[SetBeingProcessed].h_resultsA[j] << "\t X: " << (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000) << "\t Y: " << (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000) << "\t Z: " << (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000) << std::endl;
									std::cout << "AtomB: " << heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]] << "\t - Pos : " << rangeSearchSlots[SetBeingProcessed].h_resultsB[j] << "\t X: " << (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000) << "\t Y: " << (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000) << "\t Z: " << (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000) << std::endl;
									std::cout << std::endl;
								}
								else if (outputType == 4)
								{
									filePrinter.printLineToOutputFile("Atom A: ", heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]], "\t - Pos : ", rangeSearchSlots[SetBeingProcessed].h_resultsA[j], "\t X: ", (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000), "\t Y: ", (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000), "\t Z: ", (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000));
									filePrinter.printLineToOutputFile("Atom B: ", heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]], "\t - Pos : ", rangeSearchSlots[SetBeingProcessed].h_resultsB[j], "\t X: ", (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000), "\t Y: ", (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000), "\t Z: ", (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000));
									filePrinter.printLineToOutputFile("");
								}

							}
						}

					}

					//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


				}

				//swap the pointer to which set needs to be processed and loaded next - so that the previously loaded set will be processed and a new set will be loaded into the spot of the already loaded set.
				switchLoadingAndProcessingSets(SetBeingProcessed, SetBeingLoaded);


				//run bruteForce set runner



			}
			m = clock();
			print_elapsed(n, m, "run time for bruteset mini: ");
			std::cout << std::endl;
		}

		filePrinter.closeOpenFile();
		
	}




	for (int i = 0; i < 5; i++) //For each of the 5 range lengths of stored protein: Unpin memory.
	{
		currentHeldEntries = heldProteinSets.ProteinDataHolder[i].heldEntries;
		if (currentHeldEntries > 0)
		{
			gpuErrchk(hipHostUnregister(heldProteinSets.ProteinDataHolder[i].xCoordsSets));
			gpuErrchk(hipHostUnregister(heldProteinSets.ProteinDataHolder[i].yCoordsSets));
			gpuErrchk(hipHostUnregister(heldProteinSets.ProteinDataHolder[i].zCoordsSets));
			gpuErrchk(hipHostUnregister(heldProteinSets.ProteinDataHolder[i].namesSets));
		}
	}

	for (int y=0;y<2;y++)
{
	gpuErrchk(hipFree(rangeSearchSlots[y].d_xCoords));
	gpuErrchk(hipFree(rangeSearchSlots[y].d_yCoords));
	gpuErrchk(hipFree(rangeSearchSlots[y].d_zCoords));
	gpuErrchk(hipFree(rangeSearchSlots[y].d_names));

		gpuErrchk(hipFree(rangeSearchSlots[y].d_resultsCount));
		gpuErrchk(hipFree(rangeSearchSlots[y].d_resultsA));
		gpuErrchk(hipFree(rangeSearchSlots[y].d_resultsB));
		gpuErrchk(hipFree(rangeSearchSlots[y].d_elementAList));
		gpuErrchk(hipFree(rangeSearchSlots[y].d_elementBList));
		gpuErrchk(hipFree(rangeSearchSlots[y].d_aCount));
		gpuErrchk(hipFree(rangeSearchSlots[y].d_bCount));

//	gpuErrchk(hipFree(rangeSearchSlots[1].d_xCoords));
//	gpuErrchk(hipFree(rangeSearchSlots[1].d_yCoords));
//	gpuErrchk(hipFree(rangeSearchSlots[1].d_zCoords));
//	gpuErrchk(hipFree(rangeSearchSlots[1].d_names));

	hipStreamDestroy(streams[y]);
//	hipStreamDestroy(streams[1]);
}
hipDeviceReset(); 
}



void gpuBruteForceRangeSearchAllLoadedSets(rangeSearchSettings& settings, ProteinDataHandler heldProteinSets, AtomToNumHashTable atomReferenceTable)
{
	std::cout <<"PERFORMING TYPE 2 RANGE SEARCH: SINGLE BULK BATCH GPU BRUTE FORCE" << std::endl;
	//all data has been preloaded into the host side ProteinDataHandler object. All that is needed is to initialise containers, moves sets of data to the gpu, process those sets of data and then return the results.

	//Initialise host and device data holders
	gpuRangeSearchResources rangeSearch;
	int safeHolderSize = 32000 * 1200 * 4; //home gpu probably cant hold this many -.-
	//int blocks = 0;
	//int threads = 0;
	//int currentEntry = 0;
	//gpuErrchk(hipMalloc((void**)&rangeSearch.d_atomACount, sizeof(int)));
	//gpuErrchk(hipMalloc((void**)&rangeSearch.d_atomAPositionList, safeHolderSize));
	//gpuErrchk(hipMalloc((void**)&rangeSearch.d_atomACurrentSearchDimensions, safeHolderSize));
	//gpuErrchk(hipMalloc((void**)&rangeSearch.d_atomACurrentSearchKdTreePositions, safeHolderSize));
	//gpuErrchk(hipMalloc((void**)&rangeSearch.d_atomAMatches, safeHolderSize));
	//gpuErrchk(hipMalloc((void**)&rangeSearch.d_atomBMatches, safeHolderSize));
	//gpuErrchk(hipMalloc((void**)&rangeSearch.d_MatchesCount, sizeof(int)));
	//	gpuErrchk(hipMalloc((void**)&rangeSearch.d_nextSearchCount, sizeof(int)));
	//	gpuErrchk(hipMalloc((void**)&rangeSearch.d_completionFlag, sizeof(int)));
	//	gpuErrchk(hipMalloc((void**)&rangeSearch.d_kdTreeSets, safeHolderSize * 2));

	gpuErrchk(hipMalloc((void**)&rangeSearch.d_namesSets, safeHolderSize / 2));

	//gpuErrchk(hipMalloc((void**)&rangeSearch.d_xyzCoordsSets, safeHolderSize * 3));

	gpuErrchk(hipMalloc((void**)&rangeSearch.d_xCoordsSets, safeHolderSize));
	gpuErrchk(hipMalloc((void**)&rangeSearch.d_yCoordsSets, safeHolderSize));
	gpuErrchk(hipMalloc((void**)&rangeSearch.d_zCoordsSets, safeHolderSize));

	//	rangeSearch.h_atomACount = (int*)malloc(sizeof(int));
	//	rangeSearch.h_atomAPositionList = (int*)malloc(safeHolderSize);
	//	rangeSearch.h_nextSearchCount = (int*)malloc(sizeof(int));
	//	rangeSearch.h_completionFlag = (int*)malloc(sizeof(int));
	//	rangeSearch.h_MatchesCount = (int*)malloc(sizeof(int));
	//	rangeSearch.h_atomAMatches = (int*)malloc(safeHolderSize);
	//	rangeSearch.h_atomBMatches = (int*)malloc(safeHolderSize);

	//hipStream_t streams[3];


	//hipStreamCreate(&streams[0]);
	//hipStreamCreate(&streams[1]);
	//hipStreamCreate(&streams[2]);

	//load data into holder arrays:
	//int entrySize = heldProteinSets.ProteinDataHolder[0].MaxEntrySize;
	//int kdEntrySize = heldProteinSets.ProteinDataHolder[0].KdTreeSize;

	//gpuErrchk(hipMemcpyAsync(rangeSearch.d_xCoordsSets, heldProteinSets.ProteinDataHolder[0].xCoordsSets, entrySize*sizeof(int), hipMemcpyHostToDevice, streams[0]));
	//gpuErrchk(hipMemcpyAsync(rangeSearch.d_yCoordsSets, heldProteinSets.ProteinDataHolder[0].yCoordsSets, entrySize*sizeof(int), hipMemcpyHostToDevice, streams[1]));
	//gpuErrchk(hipMemcpyAsync(rangeSearch.d_zCoordsSets, heldProteinSets.ProteinDataHolder[0].zCoordsSets, entrySize*sizeof(int), hipMemcpyHostToDevice, streams[2]));
	//Leaving this here as backup notation for the kd tree search - delete when its operational
	//gpuErrchk(hipMemcpyAsync(rangeSearch.d_xyzCoordsSets, heldProteinSets.ProteinDataHolder[0].xCoordsSets, entrySize*sizeof(int), hipMemcpyHostToDevice, streams[0]));
	//gpuErrchk(hipMemcpyAsync(rangeSearch.d_xyzCoordsSets + entrySize, heldProteinSets.ProteinDataHolder[0].yCoordsSets, entrySize*sizeof(int), hipMemcpyHostToDevice, streams[1]));
	//gpuErrchk(hipMemcpyAsync(rangeSearch.d_xyzCoordsSets + entrySize * 2, heldProteinSets.ProteinDataHolder[0].zCoordsSets, entrySize*sizeof(int), hipMemcpyHostToDevice, streams[2]));
	//gpuErrchk(hipMemcpyAsync(rangeSearch.d_namesSets, heldProteinSets.ProteinDataHolder[0].namesSets, entrySize / sizeof(int)*sizeof(short), hipMemcpyHostToDevice, streams[0]));
	//	gpuErrchk(hipMemcpyAsync(rangeSearch.d_kdTreeSets, heldProteinSets.ProteinDataHolder[0].kdTrees, kdEntrySize*sizeof(int), hipMemcpyHostToDevice, streams[0]));



	int soughtAtomANumber = atomReferenceTable.retrieveHashValue(settings.AtomTypeOne);
	int soughtAtomBNumber = atomReferenceTable.retrieveHashValue(settings.AtomTypeTwo);

	//gpuErrchk(hipMemsetAsync(rangeSearch.d_atomACount, 0, sizeof(int), streams[0]));
	//gpuErrchk(hipMemsetAsync(rangeSearch.d_atomAPositionList, -1, safeHolderSize, streams[1]));
	//gpuErrchk(hipMemsetAsync(rangeSearch.d_atomAMatches, -1, safeHolderSize, streams[2]));
	//gpuErrchk(hipMemsetAsync(rangeSearch.d_atomBMatches, -1, safeHolderSize, streams[0]));
	//	gpuErrchk(hipMemsetAsync(rangeSearch.d_atomACurrentSearchDimensions, 0, safeHolderSize, streams[1]));
	//	gpuErrchk(hipMemsetAsync(rangeSearch.d_atomACurrentSearchKdTreePositions, -1, safeHolderSize, streams[2]));
	//	gpuErrchk(hipMemsetAsync(rangeSearch.d_MatchesCount, 0, sizeof(int), streams[0]));
	//	gpuErrchk(hipMemsetAsync(rangeSearch.d_nextSearchCount, 0, sizeof(int), streams[1]));
	//	gpuErrchk(hipMemsetAsync(rangeSearch.d_completionFlag, 0, sizeof(int), streams[2]));
	//	rangeSearch.h_MatchesCount[0] = 0;

	//	calculateInitialBlocksAndThreads(blocks, threads, heldProteinSets.ProteinDataHolder[0].MaxEntrySize);



	//	hipDeviceSynchronize();

	//	device_side__locateElement << <blocks, threads, 0, streams[0] >> >(rangeSearch.d_namesSets, currentEntry, soughtAtomANumber, rangeSearch.d_atomAPositionList, rangeSearch.d_atomACount, rangeSearch.d_atomACurrentSearchDimensions, rangeSearch.d_atomACurrentSearchKdTreePositions, heldProteinSets.ProteinDataHolder[0].MaxEntrySize);

	//	gpuErrchk(hipMemcpyAsync(rangeSearch.h_atomACount, rangeSearch.d_atomACount, sizeof(int), hipMemcpyDeviceToHost, streams[0]));
	//	gpuErrchk(hipMemcpyAsync(rangeSearch.h_atomAPositionList, rangeSearch.d_atomAPositionList, safeHolderSize, hipMemcpyDeviceToHost, streams[1]));
	int currentMaxEntrySize;
	int currentHeldEntries;
	for (int i = 0; i < 5; i++)
	{
		currentMaxEntrySize = heldProteinSets.ProteinDataHolder[i].MaxEntrySize;
		currentHeldEntries = heldProteinSets.ProteinDataHolder[i].heldEntries;
		if (currentHeldEntries > 0)
		{
			gpuErrchk(hipHostRegister(heldProteinSets.ProteinDataHolder[i].xCoordsSets, currentMaxEntrySize * currentHeldEntries * sizeof(int), 0));
			gpuErrchk(hipHostRegister(heldProteinSets.ProteinDataHolder[i].yCoordsSets, currentMaxEntrySize * currentHeldEntries * sizeof(int), 0));
			gpuErrchk(hipHostRegister(heldProteinSets.ProteinDataHolder[i].zCoordsSets, currentMaxEntrySize * currentHeldEntries * sizeof(int), 0));
			gpuErrchk(hipHostRegister(heldProteinSets.ProteinDataHolder[i].namesSets, currentMaxEntrySize * currentHeldEntries * sizeof(short), 0));
		}
	}

	for (int i = 0; i < 5; i++)
	{


		if (heldProteinSets.ProteinDataHolder[i].heldEntries > 0)
		{
			currentMaxEntrySize = heldProteinSets.ProteinDataHolder[i].MaxEntrySize;
			currentHeldEntries = heldProteinSets.ProteinDataHolder[i].heldEntries;
			std::cout << "Processing Range set: " << i << std::endl;
			std::cout << "Number of present entries is: " << currentHeldEntries << std::endl;
			gpuErrchk(hipMemcpy(rangeSearch.d_namesSets, heldProteinSets.ProteinDataHolder[i].namesSets, currentMaxEntrySize * currentHeldEntries * sizeof(short), hipMemcpyHostToDevice));
			gpuErrchk(hipMemcpy(rangeSearch.d_xCoordsSets, heldProteinSets.ProteinDataHolder[i].xCoordsSets, currentMaxEntrySize * currentHeldEntries * sizeof(int), hipMemcpyHostToDevice));
			gpuErrchk(hipMemcpy(rangeSearch.d_yCoordsSets, heldProteinSets.ProteinDataHolder[i].yCoordsSets, currentMaxEntrySize * currentHeldEntries * sizeof(int), hipMemcpyHostToDevice));
			gpuErrchk(hipMemcpy(rangeSearch.d_zCoordsSets, heldProteinSets.ProteinDataHolder[i].zCoordsSets, currentMaxEntrySize * currentHeldEntries * sizeof(int), hipMemcpyHostToDevice));

			//run bruteForce set runner
			clock_t n, m;
			n = clock();
			bruteForceSearchPreLoadedArraySets(rangeSearch.d_namesSets, rangeSearch.d_xCoordsSets, rangeSearch.d_yCoordsSets, rangeSearch.d_zCoordsSets, currentMaxEntrySize, soughtAtomANumber, soughtAtomBNumber, currentHeldEntries, settings, heldProteinSets.ProteinDataHolder[i].xCoordsSets, heldProteinSets.ProteinDataHolder[i].yCoordsSets, heldProteinSets.ProteinDataHolder[i].zCoordsSets, heldProteinSets.ProteinDataHolder[i].namesSets, i);
			m = clock();
			print_elapsed(n, m, "run time for bruteset mini: ");
			std::cout << std::endl;
		}

	}




	for (int i = 0; i < 5; i++)
	{

		//currentMaxEntrySize = heldProteinSets.ProteinDataHolder[i].MaxEntrySize;
		currentHeldEntries = heldProteinSets.ProteinDataHolder[i].heldEntries;
		if (currentHeldEntries > 0)
		{
			gpuErrchk(hipHostUnregister(heldProteinSets.ProteinDataHolder[i].xCoordsSets));
			gpuErrchk(hipHostUnregister(heldProteinSets.ProteinDataHolder[i].yCoordsSets));
			gpuErrchk(hipHostUnregister(heldProteinSets.ProteinDataHolder[i].zCoordsSets));
			gpuErrchk(hipHostUnregister(heldProteinSets.ProteinDataHolder[i].namesSets));
		}
	}


	gpuErrchk(hipFree(rangeSearch.d_xCoordsSets));
	gpuErrchk(hipFree(rangeSearch.d_yCoordsSets));
	gpuErrchk(hipFree(rangeSearch.d_zCoordsSets));
	gpuErrchk(hipFree(rangeSearch.d_namesSets));



}




void bruteForceSearchPreLoadedArraySets(short * d_namesSet, int*d_xValsSet, int*d_yValsSet, int*d_zValsSet, int MaximumlengthOfChains, short elementA, short  elementB, int numOfEntries, rangeSearchSettings& settings, int * h_xValsSets, int * h_yValSets, int *h_zValSets, short* h_names, int currentSet)
{

	gpuBruteForceResources resources;

	resources.h_resultsCount = (int*)malloc(1 * sizeof(int));
	resources.h_resultsA = (int*)malloc(MaximumlengthOfChains * 100 * sizeof(int));
	resources.h_resultsB = (int*)malloc(MaximumlengthOfChains * 100 * sizeof(int));
	resources.h_aCount = (int*)malloc(1 * sizeof(int));
	resources.h_bCount = (int*)malloc(1 * sizeof(int));
	resources.h_elementAList = (int*)malloc(MaximumlengthOfChains * sizeof(int)); //I used these for loading the element arrays back from the device to check what was in them, otherwise commented out.
	resources.h_elementBList = (int*)malloc(MaximumlengthOfChains * sizeof(int));
	gpuErrchk(hipMalloc((void**)&resources.d_resultsCount, 1 * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&resources.d_resultsA, MaximumlengthOfChains * 100 * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&resources.d_resultsB, MaximumlengthOfChains * 100 * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&resources.d_elementAList, MaximumlengthOfChains * 10 * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&resources.d_elementBList, MaximumlengthOfChains * 10 * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&resources.d_aCount, 1 * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&resources.d_bCount, 1 * sizeof(int)));
	gpuErrchk(hipMemset(resources.d_resultsCount, 0, 1 * sizeof(int)));
	gpuErrchk(hipMemset(resources.d_resultsA, -1, MaximumlengthOfChains * 10 * sizeof(int)));
	gpuErrchk(hipMemset(resources.d_resultsB, -1, MaximumlengthOfChains * 10 * sizeof(int)));
	gpuErrchk(hipMemset(resources.d_aCount, 0, 1 * sizeof(int)));
	gpuErrchk(hipMemset(resources.d_bCount, 0, 1 * sizeof(int)));

	//I assume these are for testing purposes
	gpuErrchk(hipMemcpy(resources.h_resultsCount, resources.d_resultsCount, 1 * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(resources.h_resultsA, resources.d_resultsA, 200 * sizeof(int), hipMemcpyDeviceToHost));

	//Configure the block and thread size based on max chain length size.
	resources.threads = 512;
	if (MaximumlengthOfChains < 513)
	{
		resources.blocks = 1;
	}
	else
	{
		resources.blocks = (MaximumlengthOfChains + resources.threads - 1) / resources.threads; //may need to check this behaves as expected.}
	}
	resources.concurrentThreads = resources.blocks*resources.threads;

	int outputType = settings.resultsPrintFormat;
	outputHandler filePrinter;
	std::string printType;

	if (outputType == 3)
	{
		printType = "_Summary";
	}
	else if (outputType == 4)
	{
		printType = "_Detailed";
	}


	if (outputType == 3 || outputType == 4)//will move into loop shortly
	{
		filePrinter.initializeOutputfile("GpuBruteResults_Range_", MaximumlengthOfChains, "_Files_", numOfEntries, printType);
	}



	//short * names = (short*)malloc(MaximumlengthOfChains*numOfEntries*sizeof(short));

	int numberOfFilesProcessed = 0;
	for (short i = 0; i < numOfEntries; i++)
	{
		gpuErrchk(hipMemset(resources.d_resultsCount, 0, 1 * sizeof(int))); //working
		gpuErrchk(hipMemset(resources.d_resultsA, -1, MaximumlengthOfChains * 100 * sizeof(int)));
		gpuErrchk(hipMemset(resources.d_resultsB, -1, MaximumlengthOfChains * 100 * sizeof(int)));
		gpuErrchk(hipMemset(resources.d_aCount, 0, 1 * sizeof(int)));
		gpuErrchk(hipMemset(resources.d_bCount, 0, 1 * sizeof(int)));
		gpuErrchk(hipMemset(resources.d_elementAList, -1, MaximumlengthOfChains * 10 * sizeof(int)));
		gpuErrchk(hipMemset(resources.d_elementBList, -1, MaximumlengthOfChains * 10 * sizeof(int)));

		resources.h_aCount[0] = 0;
		resources.h_bCount[0] = 0; //just for testing



		DeviceLoadedArrays_SingleProtein_LocateElements << <resources.blocks, resources.threads >> >(d_namesSet, resources.d_elementAList, resources.d_elementBList, elementA, elementB, resources.d_aCount, resources.d_bCount, MaximumlengthOfChains, i, resources.concurrentThreads, resources.d_resultsA, resources.d_resultsB, resources.d_resultsCount, d_xValsSet, d_yValsSet, d_zValsSet, settings.requiredProximity);
		DeviceLoadedArrays_SingleProtein_BruteForceSearch << <resources.blocks, resources.threads >> >(d_namesSet, resources.d_elementAList, resources.d_elementBList, elementA, elementB, resources.d_aCount, resources.d_bCount, MaximumlengthOfChains, i, resources.concurrentThreads, resources.d_resultsA, resources.d_resultsB, resources.d_resultsCount, d_xValsSet, d_yValsSet, d_zValsSet, settings.requiredProximity);

		gpuErrchk(hipMemcpy(resources.h_elementAList, resources.d_elementAList, MaximumlengthOfChains * sizeof(int), hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(resources.h_elementBList, resources.d_elementBList, MaximumlengthOfChains * sizeof(int), hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(resources.h_aCount, resources.d_aCount, 1 * sizeof(int), hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(resources.h_bCount, resources.d_bCount, 1 * sizeof(int), hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(resources.h_resultsCount, resources.d_resultsCount, 1 * sizeof(int), hipMemcpyDeviceToHost));


		if (outputType == 1 || outputType == 2)
			std::cout << "Number of matches in file " << i << " in set " << currentSet << " is: " << resources.h_resultsCount[0] << std::endl; 
		else if (outputType == 3 || outputType == 4)
			filePrinter.printLineToOutputFile("Number of matches in file ", i, " in set ", currentSet, "  is: ", resources.h_resultsCount[0]);

		//retrieve result arrays from device
		gpuErrchk(hipMemcpy(resources.h_resultsCount, resources.d_resultsCount, sizeof(int), hipMemcpyDeviceToHost));


		if (resources.h_resultsCount[0] < (MaximumlengthOfChains * 9)) //If there are too many results, the program can fail. Not sure what the max limit on results (relative to reserved space) is though.
		{


			gpuErrchk(hipMemcpy(resources.h_resultsA, resources.d_resultsA, resources.h_resultsCount[0] * 1 * sizeof(int), hipMemcpyDeviceToHost));
			gpuErrchk(hipMemcpy(resources.h_resultsB, resources.d_resultsB, resources.h_resultsCount[0] * 1 * sizeof(int), hipMemcpyDeviceToHost));

			if (resources.h_resultsCount[0] > 0)
			{

				//if ((resources.h_bCount[0]>0) || (resources.h_aCount[0]>0)) //just for testing
				//{

				//	int larger = resources.h_aCount[0];
				//	if (resources.h_bCount[0]>larger)
				//		larger = resources.h_bCount[0];
				//	for (int t = 0; t < larger; t++)
				//	{
				//              outputFile << h_elementAList[t]<<"\t"<<h_names[i*MaximumlengthOfChains+h_elementAList[t]] << "\t" << h_elementBList[t] << "\t"<< h_names[i*MaximumlengthOfChains+h_elementBList[t]] <<endl;
				//	}
				//}
				if (resources.h_resultsCount[0] > 0)
				{
					for (int j = 0; j < resources.h_resultsCount[0]; j++)
					{
						if (outputType == 2)
						{
							std::cout << "AtomA: " << h_names[i*MaximumlengthOfChains + resources.h_resultsA[j]] << "\t - Pos : " << resources.h_resultsA[j] << "\t X: " << (double(h_xValsSets[i*MaximumlengthOfChains + resources.h_resultsA[j]]) / 1000) << "\t Y: " << (double(h_yValSets[i*MaximumlengthOfChains + resources.h_resultsA[j]]) / 1000) << "\t Z: " << (double(h_zValSets[i*MaximumlengthOfChains + resources.h_resultsA[j]]) / 1000) << std::endl;
							std::cout << "AtomB: " << h_names[i*MaximumlengthOfChains + resources.h_resultsB[j]] << "\t - Pos : " << resources.h_resultsB[j] << "\t X: " << (double(h_xValsSets[i*MaximumlengthOfChains + resources.h_resultsB[j]]) / 1000) << "\t Y: " << (double(h_yValSets[i*MaximumlengthOfChains + resources.h_resultsB[j]]) / 1000) << "\t Z: " << (double(h_zValSets[i*MaximumlengthOfChains + resources.h_resultsB[j]]) / 1000) << std::endl;
							std::cout << std::endl;
						}
						else if (outputType == 4)
						{
							filePrinter.printLineToOutputFile("Atom A: ", h_names[i*MaximumlengthOfChains + resources.h_resultsA[j]], "\t - Pos : ", resources.h_resultsA[j], "\t X: ", (double(h_xValsSets[i*MaximumlengthOfChains + resources.h_resultsA[j]]) / 1000), "\t Y: ", (double(h_yValSets[i*MaximumlengthOfChains + resources.h_resultsA[j]]) / 1000), "\t Z: ", (double(h_zValSets[i*MaximumlengthOfChains + resources.h_resultsA[j]]) / 1000));
							filePrinter.printLineToOutputFile("Atom B: ", h_names[i*MaximumlengthOfChains + resources.h_resultsB[j]], "\t - Pos : ", resources.h_resultsB[j], "\t X: ", (double(h_xValsSets[i*MaximumlengthOfChains + resources.h_resultsB[j]]) / 1000), "\t Y: ", (double(h_yValSets[i*MaximumlengthOfChains + resources.h_resultsB[j]]) / 1000), "\t Z: ", (double(h_zValSets[i*MaximumlengthOfChains + resources.h_resultsB[j]]) / 1000));
							filePrinter.printLineToOutputFile("");
						}

					}
				}
			}
		}
		numberOfFilesProcessed++;
	}


	filePrinter.closeOpenFile();

	gpuErrchk(hipFree(resources.d_resultsCount));
	gpuErrchk(hipFree(resources.d_resultsA));
	gpuErrchk(hipFree(resources.d_resultsB));
	gpuErrchk(hipFree(resources.d_elementAList));
	gpuErrchk(hipFree(resources.d_elementBList));
	gpuErrchk(hipFree(resources.d_aCount));
	gpuErrchk(hipFree(resources.d_bCount));



};



//not currently used as combining the kernel creates a fault due to blocks not syncing before the second half.
__global__ void DeviceLoadedArrays_SingleProtein_CompleteBruteForceSearch(short * d_namesSet, int* d_elementAList, int* d_elementBList, short atomA, short atomB, int * d_aCount, int * d_bCount, int standardizedEntrySize, int entryNumber, int concurrentThreads, int*d_resultsAList, int*d_resultsBList, int*d_resultsCount, int* d_xValsSet, int* d_yValsSet, int* d_zValsSet, int requiredProximity)
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	int secondroundId = id;
	int resultsArrayInsertPosition;
	int currentDeviceName;
	int currentOffset = entryNumber*standardizedEntrySize;


	if (id == 0)
	{
		d_resultsCount[0] = 0;
		d_aCount[0] = 0;
		d_bCount[0] = 0;
	}
	__syncthreads();
	//while (id < (standardizedEntrySize +currentOffset))
	{
		currentDeviceName = d_namesSet[id + currentOffset];
		if (currentDeviceName == atomA)
		{
			resultsArrayInsertPosition = atomicAdd(&d_aCount[0], 1);
			d_elementAList[resultsArrayInsertPosition] = id;

		}
		if (currentDeviceName == atomB)
		{
			resultsArrayInsertPosition = atomicAdd(&d_bCount[0], 1);
			d_elementBList[resultsArrayInsertPosition] = id;
		}
		//	id = id + concurrentThreads;
	}
	__syncthreads();



	if (secondroundId < d_aCount[0])
	{
		int requiredProximitySquared = requiredProximity*requiredProximity;

		int insertPosition;
		short localAtomA = d_elementAList[secondroundId];

		int currentSetOffset = entryNumber*standardizedEntrySize;

		int localXCoord = d_xValsSet[currentSetOffset + localAtomA];
		int localYCoord = d_yValsSet[currentSetOffset + localAtomA];
		int localZCoord = d_zValsSet[currentSetOffset + localAtomA];

		short currentAtomB;
		int BCurrentXCoord;
		int BCurrentYCoord;
		int BCurrentZCoord;
		int distanceBetweenAtoms;

		for (int i = 0; i < d_bCount[0]; i++)
		{
			currentAtomB = d_elementBList[i];
			BCurrentXCoord = d_xValsSet[currentSetOffset + currentAtomB];
			BCurrentYCoord = d_yValsSet[currentSetOffset + currentAtomB];
			BCurrentZCoord = d_zValsSet[currentSetOffset + currentAtomB];
			distanceBetweenAtoms = ((localXCoord - BCurrentXCoord)*(localXCoord - BCurrentXCoord) + (localYCoord - BCurrentYCoord)*(localYCoord - BCurrentYCoord) + (localZCoord - BCurrentZCoord)*(localZCoord - BCurrentZCoord));

			int f = (requiredProximitySquared - distanceBetweenAtoms) / 100;

			//This loop is triggering regardless of it being right or wrong -.-
			//if (distanceBetweenAtoms < requiredProximitySquared)
			if (f > 0)
			{
				insertPosition = atomicAdd(&d_resultsCount[0], 1);
				d_resultsAList[insertPosition] = localAtomA;
				d_resultsBList[insertPosition] = currentAtomB;
			}
		}
	}
};



__global__ void DeviceLoadedArrays_SingleProtein_LocateElements(short * d_namesSet, int* d_elementAList, int* d_elementBList, short atomA, short atomB, int * d_aCount, int * d_bCount, int standardizedEntrySize, int entryNumber, int concurrentThreads, int*d_resultsAList, int*d_resultsBList, int*d_resultsCount, int* d_xValsSet, int* d_yValsSet, int* d_zValsSet, int requiredProximity)
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	int resultsArrayInsertPosition;
	int currentDeviceName;
	int currentOffset = entryNumber*standardizedEntrySize;


	if (id == 0)
	{
		d_resultsCount[0] = 0;
		d_aCount[0] = 0;
		d_bCount[0] = 0;
	}
	__syncthreads();
	//while (id < (standardizedEntrySize +currentOffset))
	{
		currentDeviceName = d_namesSet[id + currentOffset];
		if (currentDeviceName == atomA)
		{
			resultsArrayInsertPosition = atomicAdd(&d_aCount[0], 1);
			d_elementAList[resultsArrayInsertPosition] = id;

		}
		if (currentDeviceName == atomB)
		{
			resultsArrayInsertPosition = atomicAdd(&d_bCount[0], 1);
			d_elementBList[resultsArrayInsertPosition] = id;
		}
		//	id = id + concurrentThreads;
	}

};



__global__ void DeviceLoadedArrays_SingleProtein_BruteForceSearch(short * d_namesSet, int* d_elementAList, int* d_elementBList, short atomA, short atomB, int * d_aCount, int * d_bCount, int standardizedEntrySize, int entryNumber, int concurrentThreads, int*d_resultsAList, int*d_resultsBList, int*d_resultsCount, int* d_xValsSet, int* d_yValsSet, int* d_zValsSet, int requiredProximity)
{
	int secondroundId = threadIdx.x + blockDim.x * blockIdx.x;


	if (secondroundId < d_aCount[0])
	{
		int requiredProximitySquared = requiredProximity*requiredProximity;

		int insertPosition;
		short localAtomA = d_elementAList[secondroundId];

		int currentSetOffset = entryNumber*standardizedEntrySize;

		int localXCoord = d_xValsSet[currentSetOffset + localAtomA];
		int localYCoord = d_yValsSet[currentSetOffset + localAtomA];
		int localZCoord = d_zValsSet[currentSetOffset + localAtomA];

		short currentAtomB;
		int BCurrentXCoord;
		int BCurrentYCoord;
		int BCurrentZCoord;
		int distanceBetweenAtoms;

		for (int i = 0; i < d_bCount[0]; i++)
		{
			currentAtomB = d_elementBList[i];
			BCurrentXCoord = d_xValsSet[currentSetOffset + currentAtomB];
			BCurrentYCoord = d_yValsSet[currentSetOffset + currentAtomB];
			BCurrentZCoord = d_zValsSet[currentSetOffset + currentAtomB];
			distanceBetweenAtoms = ((localXCoord - BCurrentXCoord)*(localXCoord - BCurrentXCoord) + (localYCoord - BCurrentYCoord)*(localYCoord - BCurrentYCoord) + (localZCoord - BCurrentZCoord)*(localZCoord - BCurrentZCoord));

			int f = (requiredProximitySquared - distanceBetweenAtoms) / 100;

			//This loop is triggering regardless of it being right or wrong -.-
			//if (distanceBetweenAtoms < requiredProximitySquared)
			if (f > 0)
			{
				insertPosition = atomicAdd(&d_resultsCount[0], 1);
				d_resultsAList[insertPosition] = localAtomA;
				d_resultsBList[insertPosition] = currentAtomB;
			}
		}
	}
};





void hybridGpuCpuSearch(rangeSearchSettings& settings, ProteinDataHandler heldProteinSets, AtomToNumHashTable atomReferenceTable)
{
	int *atomAPositionList = (int*)malloc(sizeof(int) * 16384);
	int *atomACount = (int*)malloc(sizeof(int));
	int atomsPresent[2] = { 0, 0 };
	//all data has been preloaded into the host side ProteinDataHandler object. All that is needed is to initialise containers, moves sets of data to the gpu, process those sets of data and then return the results.
	std::cout << "PERFORMING Hybrid BRUTE FORCE RANGE SEARCH" << std::endl;

	hipStream_t streams[2];
	hipStreamCreate(&streams[0]);
	hipStreamCreate(&streams[1]);

	//Initialise host and device data holders
	int SetBeingProcessed = 0;
	int SetBeingLoaded = 1;
	gpuBruteForceSingleEntryResources rangeSearchSlots[2];
	//gpuBruteForceSingleEntryResources rangeSearchSlotB;






	int IndividualEntryHolderSize = 16390 * 4; //The memory required to hold a max size coordinate array + 6 more atoms

	for (int i = 0; i < 2; i++)
	{
		rangeSearchSlots[i].h_resultsCount = (int*)malloc(1 * sizeof(int));
		rangeSearchSlots[i].h_resultsA = (int*)malloc(IndividualEntryHolderSize * 100 * sizeof(int));
		rangeSearchSlots[i].h_resultsB = (int*)malloc(IndividualEntryHolderSize * 100 * sizeof(int));
		rangeSearchSlots[i].h_aCount = (int*)malloc(1 * sizeof(int));
		rangeSearchSlots[i].h_bCount = (int*)malloc(1 * sizeof(int));
		rangeSearchSlots[i].h_elementAList = (int*)malloc(IndividualEntryHolderSize * sizeof(int)); //I used these for loading the element arrays back from the device to check what was in them, otherwise commented out.
		rangeSearchSlots[i].h_elementBList = (int*)malloc(IndividualEntryHolderSize * sizeof(int));
		rangeSearchSlots[i].threads = 512;
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_resultsCount, 1 * sizeof(int)));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_resultsA, IndividualEntryHolderSize * 100 * sizeof(int)));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_resultsB, IndividualEntryHolderSize * 100 * sizeof(int)));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_elementAList, IndividualEntryHolderSize * 10 * sizeof(int)));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_elementBList, IndividualEntryHolderSize * 10 * sizeof(int)));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_aCount, 1 * sizeof(int)));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_bCount, 1 * sizeof(int)));
		gpuErrchk(hipMemset(rangeSearchSlots[i].d_resultsCount, 0, 1 * sizeof(int)));
		gpuErrchk(hipMemset(rangeSearchSlots[i].d_resultsA, -1, IndividualEntryHolderSize * 10 * sizeof(int)));
		gpuErrchk(hipMemset(rangeSearchSlots[i].d_resultsB, -1, IndividualEntryHolderSize * 10 * sizeof(int)));
		gpuErrchk(hipMemset(rangeSearchSlots[i].d_aCount, 0, 1 * sizeof(int)));
		gpuErrchk(hipMemset(rangeSearchSlots[i].d_bCount, 0, 1 * sizeof(int)));

		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_names, IndividualEntryHolderSize / 2));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_xCoords, IndividualEntryHolderSize));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_yCoords, IndividualEntryHolderSize));
		gpuErrchk(hipMalloc((void**)&rangeSearchSlots[i].d_zCoords, IndividualEntryHolderSize));

	}


	int soughtAtomANumber = atomReferenceTable.retrieveHashValue(settings.AtomTypeOne);
	int soughtAtomBNumber = atomReferenceTable.retrieveHashValue(settings.AtomTypeTwo);

	int currentMaxEntrySize;
	int currentHeldEntries;


	for (int i = 0; i < 5; i++) //For each of the 5 range lengths of stored protein: Pin memory.
	{
		currentMaxEntrySize = heldProteinSets.ProteinDataHolder[i].MaxEntrySize;
		currentHeldEntries = heldProteinSets.ProteinDataHolder[i].heldEntries;
		if (currentHeldEntries > 0)
		{
			gpuErrchk(hipHostRegister(heldProteinSets.ProteinDataHolder[i].xCoordsSets, currentMaxEntrySize * currentHeldEntries * sizeof(int), 0));
			gpuErrchk(hipHostRegister(heldProteinSets.ProteinDataHolder[i].yCoordsSets, currentMaxEntrySize * currentHeldEntries * sizeof(int), 0));
			gpuErrchk(hipHostRegister(heldProteinSets.ProteinDataHolder[i].zCoordsSets, currentMaxEntrySize * currentHeldEntries * sizeof(int), 0));
			gpuErrchk(hipHostRegister(heldProteinSets.ProteinDataHolder[i].namesSets, currentMaxEntrySize * currentHeldEntries * sizeof(short), 0));
		}
	}

	for (int i = 0; i < 5; i++)//For each of the 5 range lengths of stored protein:
	{
		currentMaxEntrySize = heldProteinSets.ProteinDataHolder[i].MaxEntrySize;
		int TotalEntries = heldProteinSets.ProteinDataHolder[i].heldEntries;

		clock_t n, m;

		if (currentMaxEntrySize < 513)
		{
			rangeSearchSlots[1].blocks = 1;
			rangeSearchSlots[0].blocks = 1;
		}
		else
		{
			rangeSearchSlots[1].blocks = (currentMaxEntrySize + rangeSearchSlots[1].threads - 1) / rangeSearchSlots[1].threads;
			rangeSearchSlots[0].blocks = (currentMaxEntrySize + rangeSearchSlots[1].threads - 1) / rangeSearchSlots[1].threads;
		}
		rangeSearchSlots[1].concurrentThreads = rangeSearchSlots[1].blocks*rangeSearchSlots[1].threads;
		rangeSearchSlots[0].concurrentThreads = rangeSearchSlots[0].blocks*rangeSearchSlots[0].threads;

		int outputType = settings.resultsPrintFormat;
		outputHandler filePrinter;
		std::string printType;
		if (outputType == 3)
			printType = "_Summary";
		else if (outputType == 4)
			printType = "_Detailed";

		if (outputType == 3 || outputType == 4)	{ filePrinter.initializeOutputfile("GpuBruteResults_Range_", currentMaxEntrySize, "_Files_", TotalEntries, printType); }

		n = clock();

		if (TotalEntries > 0)
		{
			std::cout << "Processing Range set: " << i << std::endl;
			std::cout << "Number of present entries is: " << heldProteinSets.ProteinDataHolder[i].heldEntries << std::endl;
			for (int currentEntry = 0; currentEntry < TotalEntries + 1; currentEntry++)
			{
				if (currentEntry == 0)
				{

					searchEntryInSecondaryPositionStructureForAtom(soughtAtomANumber, currentEntry, heldProteinSets.ProteinDataHolder[i].MaxEntrySize, heldProteinSets.ProteinDataHolder[i].compositionCountsList, heldProteinSets.ProteinDataHolder[i].compositionLists, heldProteinSets.ProteinDataHolder[i].compositionPointers, atomAPositionList, atomACount);
					if (atomACount[0]>0)
					{
						atomsPresent[SetBeingLoaded] = 1;


						//Load first set of details onto the gpu but do not process them -- needs work
						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_resultsCount, 0, 1 * sizeof(int), streams[SetBeingLoaded]));
						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_resultsA, -1, IndividualEntryHolderSize * 10 * sizeof(int), streams[SetBeingLoaded]));
						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_resultsB, -1, IndividualEntryHolderSize * 10 * sizeof(int), streams[SetBeingLoaded]));
						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_aCount, 0, 1 * sizeof(int), streams[SetBeingLoaded]));
						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_bCount, 0, 1 * sizeof(int), streams[SetBeingLoaded]));
						gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_names, heldProteinSets.ProteinDataHolder[i].namesSets + currentMaxEntrySize*currentEntry, currentMaxEntrySize * sizeof(short), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
						gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_xCoords, heldProteinSets.ProteinDataHolder[i].xCoordsSets + currentMaxEntrySize*currentEntry, currentMaxEntrySize *  sizeof(int), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
						gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_yCoords, heldProteinSets.ProteinDataHolder[i].yCoordsSets + currentMaxEntrySize*currentEntry, currentMaxEntrySize * sizeof(int), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
						gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_zCoords, heldProteinSets.ProteinDataHolder[i].zCoordsSets + currentMaxEntrySize*currentEntry, currentMaxEntrySize * sizeof(int), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
					}
					else
						atomsPresent[SetBeingLoaded] = -1;
				}
				else if (currentEntry == TotalEntries)
				{
					//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
					//process current loaded set
					if (atomsPresent[SetBeingProcessed] > 0)
					{
						rangeSearchSlots[SetBeingProcessed].h_aCount[0] = 0;
						rangeSearchSlots[SetBeingProcessed].h_bCount[0] = 0; //just for testing




						DeviceLoadedArrays_SingleProtein_LocateElements << <rangeSearchSlots[SetBeingProcessed].blocks, rangeSearchSlots[SetBeingProcessed].threads >> >(rangeSearchSlots[SetBeingProcessed].d_names, rangeSearchSlots[SetBeingProcessed].d_elementAList, rangeSearchSlots[SetBeingProcessed].d_elementBList, soughtAtomANumber, soughtAtomBNumber, rangeSearchSlots[SetBeingProcessed].d_aCount, rangeSearchSlots[SetBeingProcessed].d_bCount, currentMaxEntrySize, 0, rangeSearchSlots[SetBeingProcessed].concurrentThreads, rangeSearchSlots[SetBeingProcessed].d_resultsA, rangeSearchSlots[SetBeingProcessed].d_resultsB, rangeSearchSlots[SetBeingProcessed].d_resultsCount, rangeSearchSlots[SetBeingProcessed].d_xCoords, rangeSearchSlots[SetBeingProcessed].d_yCoords, rangeSearchSlots[SetBeingProcessed].d_zCoords, settings.requiredProximity);
						DeviceLoadedArrays_SingleProtein_BruteForceSearch << <rangeSearchSlots[SetBeingProcessed].blocks, rangeSearchSlots[SetBeingProcessed].threads >> >(rangeSearchSlots[SetBeingProcessed].d_names, rangeSearchSlots[SetBeingProcessed].d_elementAList, rangeSearchSlots[SetBeingProcessed].d_elementBList, soughtAtomANumber, soughtAtomBNumber, rangeSearchSlots[SetBeingProcessed].d_aCount, rangeSearchSlots[SetBeingProcessed].d_bCount, currentMaxEntrySize, 0, rangeSearchSlots[SetBeingProcessed].concurrentThreads, rangeSearchSlots[SetBeingProcessed].d_resultsA, rangeSearchSlots[SetBeingProcessed].d_resultsB, rangeSearchSlots[SetBeingProcessed].d_resultsCount, rangeSearchSlots[SetBeingProcessed].d_xCoords, rangeSearchSlots[SetBeingProcessed].d_yCoords, rangeSearchSlots[SetBeingProcessed].d_zCoords, settings.requiredProximity);

						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_elementAList, rangeSearchSlots[SetBeingProcessed].d_elementAList, currentMaxEntrySize * sizeof(int), hipMemcpyDeviceToHost));
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_elementBList, rangeSearchSlots[SetBeingProcessed].d_elementBList, currentMaxEntrySize * sizeof(int), hipMemcpyDeviceToHost));
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_aCount, rangeSearchSlots[SetBeingProcessed].d_aCount, 1 * sizeof(int), hipMemcpyDeviceToHost));
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_bCount, rangeSearchSlots[SetBeingProcessed].d_bCount, 1 * sizeof(int), hipMemcpyDeviceToHost));


						if (outputType == 1 || outputType == 2)
							std::cout << "Number of matches in file " << currentEntry << " in set " << i << " is: " << rangeSearchSlots[SetBeingProcessed].h_aCount[0] << std::endl;
						else if (outputType == 3 || outputType == 4)
							filePrinter.printLineToOutputFile("Number of matches in file ", currentEntry, " in set ", i, "  is: ", rangeSearchSlots[SetBeingProcessed].h_aCount[0]);

						//retrieve result arrays from device
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsCount, rangeSearchSlots[SetBeingProcessed].d_resultsCount, sizeof(int), hipMemcpyDeviceToHost));


						if (rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] < (currentMaxEntrySize * 9)) //If there are too many results, the program can fail. Not sure what the max limit on results (relative to reserved space) is though.
						{


							gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsA, rangeSearchSlots[SetBeingProcessed].d_resultsA, rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] * 2 * sizeof(int), hipMemcpyDeviceToHost));
							gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsB, rangeSearchSlots[SetBeingProcessed].d_resultsB, rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] * 2 * sizeof(int), hipMemcpyDeviceToHost));


							if (rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] > 0)
							{
								for (int j = 0; j < rangeSearchSlots[SetBeingProcessed].h_resultsCount[0]; j++)
								{
									if (outputType == 2)
									{
										std::cout << "AtomA: " << heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]] << "\t - Pos : " << rangeSearchSlots[SetBeingProcessed].h_resultsA[j] << "\t X: " << (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000) << "\t Y: " << (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000) << "\t Z: " << (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000) << std::endl;
										std::cout << "AtomB: " << heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]] << "\t - Pos : " << rangeSearchSlots[SetBeingProcessed].h_resultsB[j] << "\t X: " << (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000) << "\t Y: " << (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000) << "\t Z: " << (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000) << std::endl;
										std::cout << std::endl;
									}
									else if (outputType == 4)
									{
										filePrinter.printLineToOutputFile("Atom A: ", heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]], "\t - Pos : ", rangeSearchSlots[SetBeingProcessed].h_resultsA[j], "\t X: ", (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000), "\t Y: ", (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000), "\t Z: ", (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000));
										filePrinter.printLineToOutputFile("Atom B: ", heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]], "\t - Pos : ", rangeSearchSlots[SetBeingProcessed].h_resultsB[j], "\t X: ", (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000), "\t Y: ", (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000), "\t Z: ", (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000));
										filePrinter.printLineToOutputFile("");
									}

								}
							}

						}
						//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
					}

				}
				else
				{


					//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
					//Load next set
					//Load next set of details onto the gpu
					searchEntryInSecondaryPositionStructureForAtom(soughtAtomANumber, currentEntry, heldProteinSets.ProteinDataHolder[i].MaxEntrySize, heldProteinSets.ProteinDataHolder[i].compositionCountsList, heldProteinSets.ProteinDataHolder[i].compositionLists, heldProteinSets.ProteinDataHolder[i].compositionPointers, atomAPositionList, atomACount);
					if (atomACount[0] > 0)
					{
						atomsPresent[SetBeingLoaded] = 1;
						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_resultsCount, 0, 1 * sizeof(int), streams[SetBeingLoaded]));
						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_resultsA, -1, IndividualEntryHolderSize * 10 * sizeof(int), streams[SetBeingLoaded]));
						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_resultsB, -1, IndividualEntryHolderSize * 10 * sizeof(int), streams[SetBeingLoaded]));
						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_aCount, 0, 1 * sizeof(int), streams[SetBeingLoaded]));
						gpuErrchk(hipMemsetAsync(rangeSearchSlots[SetBeingLoaded].d_bCount, 0, 1 * sizeof(int), streams[SetBeingLoaded]));
						gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_names, heldProteinSets.ProteinDataHolder[i].namesSets + currentMaxEntrySize*currentEntry, currentMaxEntrySize * sizeof(short), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
						gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_xCoords, heldProteinSets.ProteinDataHolder[i].xCoordsSets + currentMaxEntrySize*currentEntry, currentMaxEntrySize *  sizeof(int), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
						gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_yCoords, heldProteinSets.ProteinDataHolder[i].yCoordsSets + currentMaxEntrySize*currentEntry, currentMaxEntrySize * sizeof(int), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
						gpuErrchk(hipMemcpyAsync(rangeSearchSlots[SetBeingLoaded].d_zCoords, heldProteinSets.ProteinDataHolder[i].zCoordsSets + currentMaxEntrySize*currentEntry, currentMaxEntrySize * sizeof(int), hipMemcpyHostToDevice, streams[SetBeingLoaded]));
					}
					else
						atomsPresent[SetBeingLoaded] = -1;
					//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////



					//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
					//process current loaded set
					if (atomsPresent[SetBeingProcessed] > 0)
					{
						rangeSearchSlots[SetBeingProcessed].h_aCount[0] = 0;
						rangeSearchSlots[SetBeingProcessed].h_bCount[0] = 0; //just for testing


						DeviceLoadedArrays_SingleProtein_LocateElements << <rangeSearchSlots[SetBeingProcessed].blocks, rangeSearchSlots[SetBeingProcessed].threads >> >(rangeSearchSlots[SetBeingProcessed].d_names, rangeSearchSlots[SetBeingProcessed].d_elementAList, rangeSearchSlots[SetBeingProcessed].d_elementBList, soughtAtomANumber, soughtAtomBNumber, rangeSearchSlots[SetBeingProcessed].d_aCount, rangeSearchSlots[SetBeingProcessed].d_bCount, currentMaxEntrySize, 0, rangeSearchSlots[SetBeingProcessed].concurrentThreads, rangeSearchSlots[SetBeingProcessed].d_resultsA, rangeSearchSlots[SetBeingProcessed].d_resultsB, rangeSearchSlots[SetBeingProcessed].d_resultsCount, rangeSearchSlots[SetBeingProcessed].d_xCoords, rangeSearchSlots[SetBeingProcessed].d_yCoords, rangeSearchSlots[SetBeingProcessed].d_zCoords, settings.requiredProximity);
						DeviceLoadedArrays_SingleProtein_BruteForceSearch << <rangeSearchSlots[SetBeingProcessed].blocks, rangeSearchSlots[SetBeingProcessed].threads >> >(rangeSearchSlots[SetBeingProcessed].d_names, rangeSearchSlots[SetBeingProcessed].d_elementAList, rangeSearchSlots[SetBeingProcessed].d_elementBList, soughtAtomANumber, soughtAtomBNumber, rangeSearchSlots[SetBeingProcessed].d_aCount, rangeSearchSlots[SetBeingProcessed].d_bCount, currentMaxEntrySize, 0, rangeSearchSlots[SetBeingProcessed].concurrentThreads, rangeSearchSlots[SetBeingProcessed].d_resultsA, rangeSearchSlots[SetBeingProcessed].d_resultsB, rangeSearchSlots[SetBeingProcessed].d_resultsCount, rangeSearchSlots[SetBeingProcessed].d_xCoords, rangeSearchSlots[SetBeingProcessed].d_yCoords, rangeSearchSlots[SetBeingProcessed].d_zCoords, settings.requiredProximity);

						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_elementAList, rangeSearchSlots[SetBeingProcessed].d_elementAList, currentMaxEntrySize * sizeof(int), hipMemcpyDeviceToHost));
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_elementBList, rangeSearchSlots[SetBeingProcessed].d_elementBList, currentMaxEntrySize * sizeof(int), hipMemcpyDeviceToHost));
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_aCount, rangeSearchSlots[SetBeingProcessed].d_aCount, 1 * sizeof(int), hipMemcpyDeviceToHost));
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_bCount, rangeSearchSlots[SetBeingProcessed].d_bCount, 1 * sizeof(int), hipMemcpyDeviceToHost));


						if (outputType == 1 || outputType == 2)
							std::cout << "Number of matches in file " << currentEntry << " in set " << i << " is: " << rangeSearchSlots[SetBeingProcessed].h_aCount[0] << std::endl;
						else if (outputType == 3 || outputType == 4)
							filePrinter.printLineToOutputFile("Number of matches in file ", currentEntry, " in set ", i, "  is: ", rangeSearchSlots[SetBeingProcessed].h_aCount[0]);

						//retrieve result arrays from device
						gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsCount, rangeSearchSlots[SetBeingProcessed].d_resultsCount, sizeof(int), hipMemcpyDeviceToHost));


						if (rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] < (currentMaxEntrySize * 9)) //If there are too many results, the program can fail. Not sure what the max limit on results (relative to reserved space) is though.
						{


							gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsA, rangeSearchSlots[SetBeingProcessed].d_resultsA, rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] * 2 * sizeof(int), hipMemcpyDeviceToHost));
							gpuErrchk(hipMemcpy(rangeSearchSlots[SetBeingProcessed].h_resultsB, rangeSearchSlots[SetBeingProcessed].d_resultsB, rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] * 2 * sizeof(int), hipMemcpyDeviceToHost));


							if (rangeSearchSlots[SetBeingProcessed].h_resultsCount[0] > 0)
							{
								for (int j = 0; j < rangeSearchSlots[SetBeingProcessed].h_resultsCount[0]; j++)
								{
									if (outputType == 2)
									{
										std::cout << "AtomA: " << heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]] << "\t - Pos : " << rangeSearchSlots[SetBeingProcessed].h_resultsA[j] << "\t X: " << (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000) << "\t Y: " << (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000) << "\t Z: " << (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000) << std::endl;
										std::cout << "AtomB: " << heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]] << "\t - Pos : " << rangeSearchSlots[SetBeingProcessed].h_resultsB[j] << "\t X: " << (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000) << "\t Y: " << (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000) << "\t Z: " << (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000) << std::endl;
										std::cout << std::endl;
									}
									else if (outputType == 4)
									{
										filePrinter.printLineToOutputFile("Atom A: ", heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]], "\t - Pos : ", rangeSearchSlots[SetBeingProcessed].h_resultsA[j], "\t X: ", (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000), "\t Y: ", (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000), "\t Z: ", (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsA[j]]) / 1000));
										filePrinter.printLineToOutputFile("Atom B: ", heldProteinSets.ProteinDataHolder[i].namesSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]], "\t - Pos : ", rangeSearchSlots[SetBeingProcessed].h_resultsB[j], "\t X: ", (double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000), "\t Y: ", (double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000), "\t Z: ", (double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[i*currentMaxEntrySize + rangeSearchSlots[SetBeingProcessed].h_resultsB[j]]) / 1000));
										filePrinter.printLineToOutputFile("");
									}

								}
							}

						}

						//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
					}

				}

				switchLoadingAndProcessingSets(SetBeingProcessed, SetBeingLoaded);



			}
			m = clock();
			print_elapsed(n, m, "run time for bruteset mini: ");
			std::cout << std::endl;
		}

		filePrinter.closeOpenFile();


	}


	for (int i = 0; i < 5; i++) //For each of the 5 range lengths of stored protein: Unpin memory.
	{
		currentHeldEntries = heldProteinSets.ProteinDataHolder[i].heldEntries;

		if (currentHeldEntries > 0)
		{
			gpuErrchk(hipHostUnregister(heldProteinSets.ProteinDataHolder[i].xCoordsSets));
			gpuErrchk(hipHostUnregister(heldProteinSets.ProteinDataHolder[i].yCoordsSets));
			gpuErrchk(hipHostUnregister(heldProteinSets.ProteinDataHolder[i].zCoordsSets));
			gpuErrchk(hipHostUnregister(heldProteinSets.ProteinDataHolder[i].namesSets));
		}
	}

	for (int i = 0; i < 2; i++)
	{
		gpuErrchk(hipFree(rangeSearchSlots[i].d_xCoords));
		gpuErrchk(hipFree(rangeSearchSlots[i].d_yCoords));
		gpuErrchk(hipFree(rangeSearchSlots[i].d_zCoords));
		gpuErrchk(hipFree(rangeSearchSlots[i].d_names));

		gpuErrchk(hipFree(rangeSearchSlots[i].d_resultsCount));
		gpuErrchk(hipFree(rangeSearchSlots[i].d_resultsA));
		gpuErrchk(hipFree(rangeSearchSlots[i].d_resultsB));
		gpuErrchk(hipFree(rangeSearchSlots[i].d_elementAList));
		gpuErrchk(hipFree(rangeSearchSlots[i].d_elementBList));
		gpuErrchk(hipFree(rangeSearchSlots[i].d_aCount));
		gpuErrchk(hipFree(rangeSearchSlots[i].d_bCount));
	}
	/*gpuErrchk(hipFree(rangeSearchSlots[1].d_xCoords));
	gpuErrchk(hipFree(rangeSearchSlots[1].d_yCoords));
	gpuErrchk(hipFree(rangeSearchSlots[1].d_zCoords));
	gpuErrchk(hipFree(rangeSearchSlots[1].d_names));*/

	hipStreamDestroy(streams[0]);
	hipStreamDestroy(streams[1]);
}


