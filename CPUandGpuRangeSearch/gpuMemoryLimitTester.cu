#include "gpuMemoryLimitTester.cuh"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


void populateArrayWithRandomNumbers(int *array, int size)
{
	for (int i = 0; i < size; i++)
	{
		array[i] = rand();

	}

};

void checkGpuMemoryLimit()
{
	int memoryAmount =983040000;
	int memoryIncrement=10000;

	int* host_array;
	int* device_array;

	host_array = (int *)malloc(3000 * sizeof(int));
	populateArrayWithRandomNumbers(host_array, 3000);	



	for (int i = 0; i < 300000; i++)
	{
		std::cout << "Attempting to allocate gpu memory size of: " << memoryAmount << std::endl;

		gpuErrchk(hipMalloc((void**)&device_array, memoryAmount*sizeof(int)));
	//	host_array = (int *)malloc(memoryAmount*sizeof(int));
	//	populateArrayWithRandomNumbers(host_array, memoryAmount);
		hipDeviceSynchronize();		
		gpuErrchk(hipMemcpy(device_array, host_array, 3000*sizeof(int), hipMemcpyHostToDevice));
		hipDeviceSynchronize();


		gpuErrchk(hipFree(device_array));
		std::cout << "Succesfully copied " << memoryAmount << " to the gpu" << std::endl << std::endl;
		memoryAmount = memoryAmount+memoryIncrement;
	}

};
