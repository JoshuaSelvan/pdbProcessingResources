#include "gpuMemoryLimitTester.cuh"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		std::fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


void populateArrayWithRandomNumbers(int *array, int size)
{
	for (int i = 0; i < size; i++)
	{
		array[i] = rand();
	}
};

void checkGpuMemoryLimit()
{
	int memoryAmount =30000;
	//int memoryIncrement=1000;

	int* host_array;
	int* device_array;

	host_array = (int *)malloc(3000 * sizeof(int));
	populateArrayWithRandomNumbers(host_array, 3000);


	for (int i = 0; i < 300000; i++)
	{
		std::cout << "Attempting to allocate gpu memory size of: " << memoryAmount << std::endl;

		gpuErrchk(hipMalloc((void**)&device_array, memoryAmount*sizeof(int)));
		//host_array = (int *)malloc(3000*sizeof(int));
		//populateArrayWithRandomNumbers(host_array, 3000);
		hipDeviceSynchronize();
		gpuErrchk(hipMemcpy(device_array, host_array, 3000 * sizeof(int), hipMemcpyHostToDevice));
		hipDeviceSynchronize();


		gpuErrchk(hipFree(device_array));
		std::cout << "Succesfully copied data to a: " << memoryAmount << " gpu array" << std::endl << std::endl;
		memoryAmount = memoryAmount * 2;//+ memoryIncrement;
	}

};
