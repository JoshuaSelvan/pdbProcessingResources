#include "hip/hip_runtime.h"
#include"gpuKdTreeRangeSearch.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


__global__ void doSillyAdditionsOnTheGpu(int* d_array)
{
	for (int i = 0; i < 7; i++)
	{

		d_array[i]=d_array[i]+5;
	

	}



};

__global__ void device_side__locateElement(short * d_names, int entryNo, int soughtAtomNum, int* d_soughtAtomPositionList, int* d_soughtAtomCount, int*d_kdSearchCurrentDimensionList, int*d_kdSearchCurrentTreePos, int lengthOfChain)
{
	int insertPosition;
	int atomA = soughtAtomNum;

	int id = threadIdx.x + blockDim.x * blockIdx.x;
	
	__syncthreads();
	if (id < lengthOfChain)
	{
		if (d_names[id + entryNo*lengthOfChain] == atomA)
		{
			//int tempdisplayer = id;
			insertPosition = atomicAdd(&d_soughtAtomCount[0], 1); //apparently atomicAdd returns the value of the variable being increased before it is increased.
			//int tempdisplayer2 = insertPosition;
			d_soughtAtomPositionList[insertPosition] = id;
			d_kdSearchCurrentDimensionList[insertPosition] = 0;
			d_kdSearchCurrentTreePos[insertPosition] = 0;
		}
	}

	return;
}

__global__ void viewIntArrayOnGPU(int* d_array)
{
	int x = 4;
	for (int i = 0; i < 7; i++)
	{
		x = x + 3;
	}
};

__global__ void viewShortArrayOnGPU(short* d_array)
{
	int x = 4;
	for (int i = 0; i < 7; i++)
	{
		x = x + 3;
	}
};


__global__ void viewCPUIntOnGPU(int anInt)
{
	int x = 4;
	for (int i = 0; i < 7; i++)
	{
		x = x + 3;
	}
};




//for small sets
void gpuKdTreeUnoptimisedRangeSearchAllLoadedSets(rangeSearchSettings &settings, ProteinDataHandler heldProteinSets, AtomToNumHashTable atomReferenceTable)
{
	//all data has been preloaded into the host side ProteinDataHandler object. All that is needed is to initialise containers, moves sets of data to the gpu, process those sets of data and then return the results.

	std::cout << "PERFORMING GPU BASED KD-TREE RANGE SEARCH" << std::endl;
	//Initialise host and device data holders
	gpuRangeSearchResources rangeSearch;
	int safeHolderSize = 32000 *140* sizeof(int)*3;
	
	int blocks = 1;
	int threads = 1;
	int currentEntry = 0;
	gpuErrchk(hipMalloc((void**)&rangeSearch.d_atomACount, sizeof(int)));
	gpuErrchk(hipMalloc((void**)&rangeSearch.d_atomAPositionList, safeHolderSize));
	gpuErrchk(hipMalloc((void**)&rangeSearch.d_atomACurrentSearchDimensions, safeHolderSize));
	gpuErrchk(hipMalloc((void**)&rangeSearch.d_atomACurrentSearchKdTreePositions, safeHolderSize));
	gpuErrchk(hipMalloc((void**)&rangeSearch.d_atomAMatches, safeHolderSize));
	gpuErrchk(hipMalloc((void**)&rangeSearch.d_atomBMatches, safeHolderSize));
	gpuErrchk(hipMalloc((void**)&rangeSearch.d_MatchesCount, sizeof(int)));
	gpuErrchk(hipMalloc((void**)&rangeSearch.d_nextSearchCount, sizeof(int)));
	gpuErrchk(hipMalloc((void**)&rangeSearch.d_completionFlag, sizeof(int)));
	gpuErrchk(hipMalloc((void**)&rangeSearch.d_kdTreeSets, safeHolderSize * 2));
	gpuErrchk(hipMalloc((void**)&rangeSearch.d_namesSets, safeHolderSize / sizeof(int) * sizeof(short)));
	gpuErrchk(hipMalloc((void**)&rangeSearch.d_xyzCoordsSets, safeHolderSize * 3));
	rangeSearch.h_atomACount = (int*)malloc(sizeof(int));
	//rangeSearch.h_atomACurrentSearchDimensions = (int*)malloc(safeHolderSize);
	//rangeSearch.h_atomACurrentSearchKdTreePositions = (int*)malloc(safeHolderSize);
	rangeSearch.h_atomAPositionList = (int*)malloc(safeHolderSize);
	rangeSearch.h_nextSearchCount = (int*)malloc(sizeof(int));
	rangeSearch.h_completionFlag = (int*)malloc(sizeof(int));
	rangeSearch.h_MatchesCount = (int*)malloc(sizeof(int));
	rangeSearch.h_atomAMatches = (int*)malloc(safeHolderSize);
	rangeSearch.h_atomBMatches = (int*)malloc(safeHolderSize);

	hipStream_t streams[3];

	hipStreamCreate(&streams[0]);
	hipStreamCreate(&streams[1]);
	hipStreamCreate(&streams[2]);

	//load data into holder arrays:
	int entrySize;
	int kdEntrySize; 
	int proteinsInSet;


	//initiate search


	//int soughtAtomANumber = 1414;
	//int soughtAtomBNumber = 1514;

	int soughtAtomANumber = atomReferenceTable.retrieveHashValue(settings.AtomTypeOne);//heldProteinSets.ProteinDataHolder[0].namesSets[0];//
	int soughtAtomBNumber = atomReferenceTable.retrieveHashValue(settings.AtomTypeTwo);//heldProteinSets.ProteinDataHolder[0].namesSets[1];//
	int maxDistanceSquared = settings.requiredProximity*settings.requiredProximity;
	

		for (int i = 0; i < 5; i++)
	{
		entrySize = heldProteinSets.ProteinDataHolder[i].MaxEntrySize;
		kdEntrySize = heldProteinSets.ProteinDataHolder[i].KdTreeSize;
		proteinsInSet = heldProteinSets.ProteinDataHolder[i].heldEntries;
		hipHostRegister(heldProteinSets.ProteinDataHolder[i].xyzCoordsSets, proteinsInSet*entrySize*sizeof(int) * 3, 0);
		hipHostRegister(heldProteinSets.ProteinDataHolder[i].namesSets, proteinsInSet*entrySize *sizeof(short), 0);
		hipHostRegister(heldProteinSets.ProteinDataHolder[i].kdTrees, proteinsInSet*kdEntrySize*sizeof(int), 0);
	}
	
	

	for (int i = 0; i < 5; i++)
	{
		entrySize = heldProteinSets.ProteinDataHolder[i].MaxEntrySize;
		kdEntrySize = heldProteinSets.ProteinDataHolder[i].KdTreeSize;
		proteinsInSet = heldProteinSets.ProteinDataHolder[i].heldEntries;
		//the x,y and z data is loaded from the 3 host arrays into a single gpu array - the xyz array
		//gpuErrchk(hipMemcpyAsync(rangeSearch.d_xyzCoordsSets, heldProteinSets.ProteinDataHolder[i].xCoordsSets, entrySize*sizeof(int), hipMemcpyHostToDevice/*, streams[0]*/)); //seeing as the entire x lot is put before the entire y lot, i could probably still use this approach.
		//gpuErrchk(hipMemcpyAsync(rangeSearch.d_xyzCoordsSets + entrySize, heldProteinSets.ProteinDataHolder[i].yCoordsSets, entrySize*sizeof(int), hipMemcpyHostToDevice/*, streams[1]*/));
		//gpuErrchk(hipMemcpyAsync(rangeSearch.d_xyzCoordsSets + entrySize * 2, heldProteinSets.ProteinDataHolder[i].zCoordsSets, entrySize*sizeof(int), hipMemcpyHostToDevice/*, streams[2]*/));
			if (heldProteinSets.ProteinDataHolder[i].heldEntries>0)
		{
std::cout<< proteinsInSet*entrySize*sizeof(int) * 3<<std::endl;	
	gpuErrchk(hipMemcpy(rangeSearch.d_xyzCoordsSets, heldProteinSets.ProteinDataHolder[i].xyzCoordsSets, proteinsInSet*entrySize*sizeof(int) * 3, hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpyAsync(rangeSearch.d_namesSets, heldProteinSets.ProteinDataHolder[i].namesSets, proteinsInSet*entrySize *sizeof(short), hipMemcpyHostToDevice/*, streams[0]*/));
		gpuErrchk(hipMemcpyAsync(rangeSearch.d_kdTreeSets, heldProteinSets.ProteinDataHolder[i].kdTrees, proteinsInSet*kdEntrySize*sizeof(int), hipMemcpyHostToDevice/*, streams[0]*/));

		std::cout << "Processing set: " << i << " on the gpu" << std::endl;
		for (int currentEntry = 0; currentEntry < heldProteinSets.ProteinDataHolder[i].heldEntries; currentEntry++)
		{
			gpuErrchk(hipMemsetAsync(rangeSearch.d_atomACount, 0, sizeof(int)/*, streams[0]*/));
			gpuErrchk(hipMemsetAsync(rangeSearch.d_atomAPositionList, -1, safeHolderSize/*, streams[1]*/));
			gpuErrchk(hipMemsetAsync(rangeSearch.d_atomAMatches, -1, safeHolderSize/*, streams[2]*/));
			gpuErrchk(hipMemsetAsync(rangeSearch.d_atomBMatches, -1, safeHolderSize/*, streams[0]*/));
			gpuErrchk(hipMemsetAsync(rangeSearch.d_atomACurrentSearchDimensions, 0, safeHolderSize/*, streams[1]*/));
			gpuErrchk(hipMemsetAsync(rangeSearch.d_atomACurrentSearchKdTreePositions, -1, safeHolderSize/*, streams[2]*/));
			gpuErrchk(hipMemsetAsync(rangeSearch.d_MatchesCount, 0, sizeof(int)/*, streams[0]*/));
			gpuErrchk(hipMemsetAsync(rangeSearch.d_nextSearchCount, 0, sizeof(int)/*, streams[1]*/));
			gpuErrchk(hipMemsetAsync(rangeSearch.d_completionFlag, 0, sizeof(int)/*, streams[2]*/));
			rangeSearch.h_MatchesCount[0] = 0;


			hipDeviceSynchronize();
			threads = 512;
			blocks = heldProteinSets.ProteinDataHolder[i].MaxEntrySize/512;
			device_side__locateElement << <blocks, threads/*, 0, streams[0] */>> >(rangeSearch.d_namesSets, currentEntry, soughtAtomANumber, rangeSearch.d_atomAPositionList, rangeSearch.d_atomACount, rangeSearch.d_atomACurrentSearchDimensions, rangeSearch.d_atomACurrentSearchKdTreePositions, heldProteinSets.ProteinDataHolder[i].MaxEntrySize);

			gpuErrchk(hipMemcpyAsync(rangeSearch.h_atomACount, rangeSearch.d_atomACount, sizeof(int), hipMemcpyDeviceToHost, streams[0]));
			//gpuErrchk(hipMemcpyAsync(rangeSearch.h_atomAPositionList, rangeSearch.d_atomAPositionList, safeHolderSize, hipMemcpyDeviceToHost, streams[1]));
			std::cout << "File: " << currentEntry << " - atom A count: " << rangeSearch.h_atomACount[0] << "  ";
			
			int maxNumberoFCycles = 13 + (heldProteinSets.ProteinDataHolder[i].MaxEntrySize) / 1024;
			rangeSearch.blocks = 1;
			rangeSearch.threads = 1;
			if (rangeSearch.h_atomACount[0] > 0)
			{
				//std::cout << std::endl << "Entry " << currentEntry << " Contains : " << rangeSearch.h_atomACount[0] << " unique instances of element : " << soughtAtomANumber << std::endl;// If any searches are being performed on the kd tree, this prints out how many instances of the first atom are present in the tree.
				for (int p = 0; p < maxNumberoFCycles; p++)
				{
					//gpuErrchk(hipMemcpy(rangeSearch.h_atomAPositionList, rangeSearch.d_atomAPositionList, sizeof(int) * rangeSearch.h_atomACount[0], hipMemcpyDeviceToHost));// for testing only



					gpuErrchk(hipMemcpy(rangeSearch.h_atomACount, rangeSearch.d_atomACount, sizeof(int) * 1, hipMemcpyDeviceToHost));
					//std::cout << "Current active search count: " << rangeSearch.h_atomACount[0] << std::endl; //This prints how many searches are currently being proccessed on the current kd-tree

					//calculateInitialBlocksAndThreads(blocks, threads, heldProteinSets.ProteinDataHolder[i].MaxEntrySize);

					if (rangeSearch.h_atomACount[0] < 513)
					{
						rangeSearch.blocks = 1;
						rangeSearch.threads = rangeSearch.h_atomACount[0];
					}
					else
					{
						rangeSearch.threads = 512;
						rangeSearch.blocks = rangeSearch.h_atomACount[0] / 512 + 1;
					}




					gpuErrchk(hipMemcpy(rangeSearch.h_MatchesCount, rangeSearch.d_MatchesCount, sizeof(int), hipMemcpyDeviceToHost));//For testing...
					device_side_ProcessCurrentTreePositionsV3 << <rangeSearch.blocks, rangeSearch.threads >> >(rangeSearch.d_xyzCoordsSets, rangeSearch.d_namesSets, rangeSearch.d_kdTreeSets, rangeSearch.d_atomAMatches, rangeSearch.d_atomBMatches, rangeSearch.d_MatchesCount, rangeSearch.d_atomACurrentSearchKdTreePositions, rangeSearch.d_atomAPositionList, rangeSearch.d_atomACurrentSearchDimensions, rangeSearch.d_atomACount, heldProteinSets.ProteinDataHolder[i].MaxEntrySize, maxDistanceSquared, soughtAtomBNumber, rangeSearch.d_nextSearchCount, currentEntry, heldProteinSets.ProteinDataHolder[i].KdTreeSize, settings.requiredProximity, currentEntry);





					gpuErrchk(hipMemcpy(rangeSearch.h_MatchesCount, rangeSearch.d_MatchesCount, sizeof(int), hipMemcpyDeviceToHost));//For testing...
					SetNextCountAsCurrentAndCheckFlag << <1, 1 >> >(rangeSearch.d_atomACount, rangeSearch.d_nextSearchCount, rangeSearch.d_completionFlag); //update device side counters reflecting how many search items exist in the list


					gpuErrchk(hipMemcpy(rangeSearch.h_completionFlag, rangeSearch.d_completionFlag, sizeof(int), hipMemcpyDeviceToHost));
					if (rangeSearch.h_completionFlag[0] == 1)
					{
						p = 1000;
					}

				}
			}
			
			gpuErrchk(hipMemcpy(rangeSearch.h_MatchesCount, rangeSearch.d_MatchesCount, sizeof(int), hipMemcpyDeviceToHost));
			if ((rangeSearch.h_MatchesCount[0] > 0) && (rangeSearch.h_atomACount[0] > 0))
			{
				
				std::cout << "Matches in file: "<< rangeSearch.h_MatchesCount[0] << std::endl;
				gpuErrchk(hipMemcpy(rangeSearch.h_atomAMatches, rangeSearch.d_atomAMatches, sizeof(int)*rangeSearch.h_MatchesCount[0], hipMemcpyDeviceToHost));
				gpuErrchk(hipMemcpy(rangeSearch.h_atomBMatches, rangeSearch.d_atomBMatches, sizeof(int)*rangeSearch.h_MatchesCount[0], hipMemcpyDeviceToHost));
				//std::cout << std::endl << "Matches in file " << currentEntry << std::endl;
				//std::cout << "Number of matches found: " << rangeSearch.h_MatchesCount[0] << std::endl << "-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_" << std::endl;
				int valuesOffset = heldProteinSets.ProteinDataHolder[i].MaxEntrySize * 3 * currentEntry;
				for (int j = 0; j < rangeSearch.h_MatchesCount[0]; j++)
				{
				//	std::cout << "Atom A: " << heldProteinSets.ProteinDataHolder[i].namesSets[heldProteinSets.ProteinDataHolder[i].MaxEntrySize * currentEntry + rangeSearch.h_atomAMatches[j]] << "\t - Pos : " << rangeSearch.h_atomAMatches[j] << "\t X: " << ((double(heldProteinSets.ProteinDataHolder[i].xyzCoordsSets[valuesOffset + rangeSearch.h_atomAMatches[j]])) / 1000) << "\t Y: " << ((double(heldProteinSets.ProteinDataHolder[i].xyzCoordsSets[valuesOffset + heldProteinSets.ProteinDataHolder[i].MaxEntrySize + rangeSearch.h_atomAMatches[j]])) / 1000) << "\t Z: " << ((double(heldProteinSets.ProteinDataHolder[i].xyzCoordsSets[valuesOffset + heldProteinSets.ProteinDataHolder[i].MaxEntrySize * 2 + rangeSearch.h_atomAMatches[j]])) / 1000) << std::endl;
				//	std::cout << "Atom B: " << heldProteinSets.ProteinDataHolder[i].namesSets[heldProteinSets.ProteinDataHolder[i].MaxEntrySize * currentEntry + rangeSearch.h_atomBMatches[j]] << "\t - Pos : " << rangeSearch.h_atomBMatches[j] << "\t X: " << ((double(heldProteinSets.ProteinDataHolder[i].xyzCoordsSets[valuesOffset + rangeSearch.h_atomBMatches[j]])) / 1000) << "\t Y: " << ((double(heldProteinSets.ProteinDataHolder[i].xyzCoordsSets[valuesOffset + heldProteinSets.ProteinDataHolder[i].MaxEntrySize + rangeSearch.h_atomBMatches[j]])) / 1000) << "\t Z: " << ((double(heldProteinSets.ProteinDataHolder[i].xyzCoordsSets[valuesOffset + heldProteinSets.ProteinDataHolder[i].MaxEntrySize * 2 + rangeSearch.h_atomBMatches[j]])) / 1000) << std::endl << std::endl;
		//temp outtake		//	std::cout << "Atom A: " << heldProteinSets.ProteinDataHolder[i].namesSets[heldProteinSets.ProteinDataHolder[i].MaxEntrySize * currentEntry + rangeSearch.h_atomAMatches[j]] << "\t - Pos : " << rangeSearch.h_atomAMatches[j] << "\t X: " << ((double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[valuesOffset + rangeSearch.h_atomAMatches[j]])) / 1000) << "\t Y: " << ((double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[valuesOffset + heldProteinSets.ProteinDataHolder[i].MaxEntrySize + rangeSearch.h_atomAMatches[j]])) / 1000) << "\t Z: " << ((double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[valuesOffset + heldProteinSets.ProteinDataHolder[i].MaxEntrySize * 2 + rangeSearch.h_atomAMatches[j]])) / 1000) << std::endl;
				//	std::cout << "Atom B: " << heldProteinSets.ProteinDataHolder[i].namesSets[heldProteinSets.ProteinDataHolder[i].MaxEntrySize * currentEntry + rangeSearch.h_atomBMatches[j]] << "\t - Pos : " << rangeSearch.h_atomBMatches[j] << "\t X: " << ((double(heldProteinSets.ProteinDataHolder[i].xCoordsSets[valuesOffset + rangeSearch.h_atomBMatches[j]])) / 1000) << "\t Y: " << ((double(heldProteinSets.ProteinDataHolder[i].yCoordsSets[valuesOffset + heldProteinSets.ProteinDataHolder[i].MaxEntrySize + rangeSearch.h_atomBMatches[j]])) / 1000) << "\t Z: " << ((double(heldProteinSets.ProteinDataHolder[i].zCoordsSets[valuesOffset + heldProteinSets.ProteinDataHolder[i].MaxEntrySize * 2 + rangeSearch.h_atomBMatches[j]])) / 1000) << std::endl << std::endl;

				}
				//std::cout << "-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_" << std::endl;
			}
			else
				std::cout << "Matches in file: 0" << std::endl;
		}
		}

	}
	//Need to finish moving stuff here.

}





//this could use some work
void calculateInitialBlocksAndThreads(int &blocks, int&threads, int maxEntrySize)
{
	
	if (maxEntrySize > 512)
	{
		threads = 512;
		blocks = maxEntrySize / threads;
	}
}



__global__ void device_side_ProcessCurrentTreePositionsV3(int* d_xyzValues, short* d_Names, int*  d_kdSetArray, int* d_ViableElementAPairs, int* d_ViableElementBPairs, int* d_ViableElementPairCount, int* d_currentSearchLocations, int*d_currentSearchAElementPos, int* d_currentSearchDimensions, int* CurrentSearchCount, int SizeOfDimensionArray, int maxDistanceSquared, int elementB, int*d_nextSearchCount, int entryNum, int sizeOfKdTree, int maxDistance, int entryBeingWorkedOn)
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	/*Variables associated with pointing to correct dataset in large storage array*/
	///////////////////////////
	int currentSetOffset = SizeOfDimensionArray * 3 * entryNum;//As the xyzSet array holds the entries for every single xyz set being processed, the starting position of the current set needs to be added to the equation
	int kdTreeOffSet = sizeOfKdTree*entryNum;
	///////////////////////////


	int threadNamePos = d_currentSearchAElementPos[id]; //This provides the kd position of the A element that all compared items in the search will be measured to.
	int threadkdPos = d_currentSearchLocations[id]; //The position in the kdArray currently being pointed to -- does not include the kdTreeSet offset
	int currentDim = d_currentSearchDimensions[id]; //The dimension of the array, used to choose which dimension to sort by. Queued child processes will recieve currentDim+1

	int currentOffset = SizeOfDimensionArray; //Used for hoping from one dimension in the xyz array to the next -- does not include the xyzSet offset
	int currentLeftChildNode = 0; //Left child node of the current kdTree node -- not yet set;
	int currentRightChildNode = 0; //Left child node of the current kdTree node  -- not yet set ;
	double currentSquaredDistance = 0; //Distance between current node and the sought after node  -- not yet set.
	bool leftChildValid = false; //used to check if left child is worth pursuing
	bool rightChildValid = false; //used to check if right child is worth pursuing
	int ValidChildsOnThisStep = 0; //Maths allows the removal of if statements.
	int resultLocation = id; // This variable acts as a local place holder of the write destination of found matches
	int zOffset = SizeOfDimensionArray * 2; // when switching between the start location of the x,y and z arrays in the global array, y is at SizeOfDimensionArray and z at SizeOfDimensionArray * 2
	int nextSearchSavePosition;

	float elementA_xyz[3]; //Fixed values of the focused element A
	elementA_xyz[0] = d_xyzValues[currentSetOffset + threadNamePos];
	elementA_xyz[1] = d_xyzValues[currentSetOffset + threadNamePos + SizeOfDimensionArray];
	elementA_xyz[2] = d_xyzValues[currentSetOffset + threadNamePos + zOffset];


	float threadxyz[3]; //Holder for the child nodes in the kd tree being looked at. Cuts down on memory calls.
	int currentKdTreePos;

	__syncthreads();


	for (int r = 0; r < 25; r++) //limit is to ensure no more then 14 resulting searches are produced - though no tree is likely to have 25 levels
	{
		currentKdTreePos = d_kdSetArray[kdTreeOffSet + threadkdPos];
		if ((threadkdPos < sizeOfKdTree) && (currentKdTreePos > -1) && (threadkdPos > -1) && (id < CurrentSearchCount[0])) //checking that we aren't looking further then the tree actually extends. 
		{
			
			threadxyz[0] = d_xyzValues[currentSetOffset + currentKdTreePos];  //Loading the current kd position's coords into the thread
			threadxyz[1] = d_xyzValues[currentSetOffset + currentKdTreePos + SizeOfDimensionArray];
			threadxyz[2] = d_xyzValues[currentSetOffset + currentKdTreePos + zOffset];


			currentLeftChildNode = threadkdPos * 2 + 1;
			currentRightChildNode = threadkdPos * 2 + 2;

			currentSquaredDistance = pow((elementA_xyz[0] - threadxyz[0]), 2) + pow((elementA_xyz[1] - threadxyz[1]), 2) + pow((elementA_xyz[2] - threadxyz[2]), 2); //3 dimensional distance between current and target point

			if (currentSquaredDistance <= maxDistanceSquared)//This section executes if the atoms are close enough for a potential match
			{
				if (d_Names[SizeOfDimensionArray*entryNum + currentKdTreePos] == elementB)//checking if the atom is the sought partner in the pair.
				{
					resultLocation = atomicAdd(&d_ViableElementPairCount[0], 1);
					d_ViableElementAPairs[resultLocation] = threadNamePos;
					d_ViableElementBPairs[resultLocation] = currentKdTreePos;
				}




				if (currentLeftChildNode < sizeOfKdTree)//If atoms are close enough for a match to happen, both child nodes automatically need to be explored - if there are child nodes
				{
					if (currentRightChildNode < sizeOfKdTree)
					{
						nextSearchSavePosition = atomicAdd(&d_nextSearchCount[0], 1);

						d_currentSearchLocations[nextSearchSavePosition] = currentRightChildNode;
						d_currentSearchAElementPos[nextSearchSavePosition] = threadNamePos;
						d_currentSearchDimensions[nextSearchSavePosition] = (currentDim + 1) % 3;
					}
					threadkdPos = currentLeftChildNode;
					currentDim = (currentDim + 1) % 3;
				}
				else if (currentRightChildNode < sizeOfKdTree)
				{
					threadkdPos = currentRightChildNode;
					currentDim = (currentDim + 1) % 3;
				}
				else //If both child nodes return invalid, we have reached the end of the search of this branch.
				{
					return;
				}
			}
			else
			{
				ValidChildsOnThisStep = 0;
				leftChildValid = 0;
				rightChildValid = 0;
				int targetElementInCurrentDim = elementA_xyz[currentDim];
				int currentThreadxyz = threadxyz[currentDim];
				int absoluteDistanceBetweenCurrentAndTargetNode = abs(currentThreadxyz - targetElementInCurrentDim);
				bool currentNodeLessThenTargetInCurrentDim = (currentThreadxyz < targetElementInCurrentDim);
				bool  currentNodeGreaterThenTargetInCurrentDim = (currentThreadxyz > targetElementInCurrentDim);

				int d_kdSetArrayLeftChildNode = d_kdSetArray[kdTreeOffSet + currentLeftChildNode];
				int currentNodeOutsideRequiredRangeInGivenDimension = (absoluteDistanceBetweenCurrentAndTargetNode > maxDistance);
				int currentD_xyzValues;
				if (d_kdSetArrayLeftChildNode > -1)
				{
					currentD_xyzValues = d_xyzValues[currentSetOffset + d_kdSetArrayLeftChildNode + currentOffset*currentDim];
					int currentLeftChildNodeFurtherFromTargetThenCurrentNodeInGivenDimension = (abs(currentD_xyzValues - targetElementInCurrentDim) > absoluteDistanceBetweenCurrentAndTargetNode);
					int currentLeftNodeInSameDirectionAsCurrentNodeFromTargetNode = ((currentNodeLessThenTargetInCurrentDim && (currentD_xyzValues<targetElementInCurrentDim)) || (currentNodeGreaterThenTargetInCurrentDim && (currentD_xyzValues>targetElementInCurrentDim)));
					int AllConditionsMetForLeftNodeInvalid = currentNodeOutsideRequiredRangeInGivenDimension*currentLeftChildNodeFurtherFromTargetThenCurrentNodeInGivenDimension*currentLeftNodeInSameDirectionAsCurrentNodeFromTargetNode; //These can be put straight into the booleans if they confirmed to work;
					leftChildValid = 1 - AllConditionsMetForLeftNodeInvalid;
				}

				int d_kdSetArrayRightChildNode = d_kdSetArray[kdTreeOffSet + currentRightChildNode];
				if (d_kdSetArrayRightChildNode > -1)
				{
					currentD_xyzValues = d_xyzValues[currentSetOffset + d_kdSetArrayRightChildNode + currentOffset*currentDim];
					int currentRightChildNodeFurtherFromTargetThenCurrentNodeInGivenDimension = (abs(currentD_xyzValues - targetElementInCurrentDim) > absoluteDistanceBetweenCurrentAndTargetNode);
					int currentRightNodeInSameDirectionAsCurrentNodeFromTargetNode = ((currentNodeLessThenTargetInCurrentDim && (currentD_xyzValues<targetElementInCurrentDim)) || (currentNodeGreaterThenTargetInCurrentDim && (currentD_xyzValues>targetElementInCurrentDim)));
					int AllConditionsMetForRightNodeInvalid = currentNodeOutsideRequiredRangeInGivenDimension*currentRightChildNodeFurtherFromTargetThenCurrentNodeInGivenDimension*currentRightNodeInSameDirectionAsCurrentNodeFromTargetNode; //These can be put straight into the booleans if they confirmed to work;
					rightChildValid = 1 - AllConditionsMetForRightNodeInvalid;
				}


				ValidChildsOnThisStep = 0 + leftChildValid + rightChildValid;

				if (ValidChildsOnThisStep == 0)
				{
					return; //End Point for thread.
				}
				else if (ValidChildsOnThisStep == 1)
				{
					currentDim = (currentDim + 1) % 3;

					if (leftChildValid == true)
					{
						threadkdPos = currentLeftChildNode;
					}
					else
					{
						threadkdPos = currentRightChildNode;
					}
				}
				else
				{
					nextSearchSavePosition = atomicAdd(&d_nextSearchCount[0], 1);
					d_currentSearchLocations[nextSearchSavePosition] = currentRightChildNode;
					d_currentSearchAElementPos[nextSearchSavePosition] = threadNamePos;
					d_currentSearchDimensions[nextSearchSavePosition] = (currentDim + 1) % 3;

					threadkdPos = currentLeftChildNode;
					currentDim = (currentDim + 1) % 3;
				}
			}

		}
		else
		{
			return;
		}
	}




	return;
}


__global__ void SetNextCountAsCurrentAndCheckFlag(int* d_currentSearchCount, int* d_nextSearchCount, int* d_completionFlag)
{
	if (d_nextSearchCount[0] == 0)
	{
		d_completionFlag[0] = 1;
	}
	else
	{
		d_currentSearchCount[0] = d_nextSearchCount[0];
		d_nextSearchCount[0] = 0;
	}
	return;
}
